#include "hip/hip_runtime.h"
﻿

#include "iostream"
#include "vector"
#include "algorithm"
#include "cmath"
#include "chrono"
#include "GPU_parameters.h"
#include "underdtw.cuh"
#include "dtw_511.cuh"
#include <hiprand/hiprand_kernel.h>
#include "ECG_511_EARLY_EXIT_DTW.cuh"
using namespace std;
#define DOUBLE_BIAS 6
#define BIAS 3
#define MIN(x,y) ((x)<(y)?(x):(y))

__device__ void atomicMinFloat(FLOAT*  address, FLOAT val) {
        unsigned int* addr_as_ui = (unsigned int*)address;
        unsigned int old = *addr_as_ui, assumed;
        do {
            assumed = old;
            FLOAT current_val = __int_as_float(assumed);
            if (val >= current_val) break; 
            old = atomicCAS(addr_as_ui, assumed, __float_as_int(val));
        } while (assumed != old);
    }

__global__ void
GLOBAL_DIAG(int minlag, int subcount, int subseqLen, int len, int warpmax, const FLOAT *a, const FLOAT *mu,
            const FLOAT *sumU_sumL, const FLOAT *invsig, const FLOAT *norm_U_plus_norm_L_trans,
            const FLOAT *dr_bwdU_plus_dr_bwdL, const FLOAT *dc_bwd, const FLOAT *dr_fwdU_plus_dr_fwdL,
            const FLOAT *dc_fwd, const FLOAT *UTS, const FLOAT *LTS, const FLOAT *UTS_global,
            const FLOAT *LTS_global, const FLOAT *MASK_global, const FLOAT *TS2, bool *d_lb_vector,
            FLOAT *d_lb_vector_new, const FLOAT *sumMASK_global, const FLOAT *sumU_sumL_global,
            const FLOAT *dr_bwdU_plus_dr_bwdL_global, const FLOAT *dr_fwdU_plus_dr_fwdL_global,
            const FLOAT *dc_bwd_global, const FLOAT *dc_fwd_global, const FLOAT *dr_bwdMASK_global,
            const FLOAT *dr_fwdMASK_global, const FLOAT *dc_bwdTS2_global, const FLOAT *dc_fwdTS2_global,
            const FLOAT *DUL2_global, const FLOAT *DUL_global, const FLOAT *norm_U_plus_norm_L_global,
            FLOAT **my_subs, FLOAT **special_shared_vector, const FLOAT *d_bsf_global, const FLOAT *DUL_fast,
            const FLOAT *DUL2_fast, int diag, int start_pos, int end_pos)
{

    int tid = threadIdx.x + blockDim.x*blockIdx.x;

    diag = diag + tid;
    end_pos = MIN(end_pos, subcount - diag + 1);
    if (start_pos > subcount - diag + 1) {
        return;
    }

    bool *lb_vector = &d_lb_vector[tid*STEP_LENGTH];
    FLOAT *lb_vector_new = &d_lb_vector_new[tid*STEP_LENGTH];

    FLOAT bsf = *d_bsf_global;

    __syncthreads();

    FLOAT cnt_of_purn = 0;
    
    diag_fast_segment(a, subseqLen, diag, UTS, LTS, mu, sumU_sumL, invsig, norm_U_plus_norm_L_trans,
              lb_vector, dr_bwdU_plus_dr_bwdL, dc_bwd, dr_fwdU_plus_dr_fwdL, dc_fwd,
              cnt_of_purn,start_pos,end_pos,bsf,DUL_fast,DUL2_fast);

    diag_mask_global(a, subseqLen, diag, lb_vector, mu, UTS_global, LTS_global,
                     MASK_global, TS2,sumMASK_global, invsig, sumU_sumL_global,
                     dr_bwdU_plus_dr_bwdL_global, dr_fwdU_plus_dr_fwdL_global, dc_bwd_global,
                     dc_fwd_global, dr_bwdMASK_global, dr_fwdMASK_global,
                     dc_bwdTS2_global, dc_fwdTS2_global, DUL2_global, lb_vector_new, bsf,
                     my_subs,special_shared_vector, cnt_of_purn, DUL_global, norm_U_plus_norm_L_global,
                     start_pos,end_pos);

}