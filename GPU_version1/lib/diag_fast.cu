#include "hip/hip_runtime.h"

#include "iostream"
#include "ctime"
#include "vector"
#include "cmath"
#include "matrix.cuh"
#include "elseoperation.cuh"
#include "GPU_parameters.h"
#include "algorithm"
#include "chrono"
using namespace std;

__device__ void diag_fast_segment(const FLOAT *ts, int subseqlen, int diag_ID, const FLOAT *UTS, const FLOAT *LTS,
                                  const FLOAT *mu, const FLOAT *sumU_sumL, const FLOAT *invsig,
                                  const FLOAT *norm_U_plus_norm_L_trans, bool *lb_vector,
                                  const FLOAT *dr_bwdU_plus_dr_bwdL, const FLOAT *dc_bwd,
                                  const FLOAT *dr_fwdU_plus_dr_fwdL, const FLOAT *dc_fwd, FLOAT &cnt,
                                  int start_pos, int end_pos, FLOAT bsf,
                                  const FLOAT *DUL, const FLOAT *DUL2) {

    int diag = diag_ID;
    FLOAT  lb;
    FLOAT  cov_U_plus_cov_L_fir = elementWiseMultiply_p_plus_sum(ts + start_pos + diag - 1,
                                                                UTS+ start_pos, LTS+ start_pos, subseqlen);
    FLOAT  cov_U_plus_cov_L_sec = (elementWiseMultiply_p_plus_sum(ts+ start_pos , UTS + start_pos + diag - 1,
                                                                  LTS + start_pos +  diag - 1, subseqlen));
    FLOAT fir_basic = cov_U_plus_cov_L_fir;
    FLOAT sec_basic = cov_U_plus_cov_L_sec;

    int row = start_pos;
    int col = start_pos + diag - 1;
    FLOAT  tt = (cov_U_plus_cov_L_fir - mu[col] * sumU_sumL[row]) * invsig[col];
    FLOAT  local_del = norm_U_plus_norm_L_trans[row] - 2 * tt;

    lb = 0.5*(sqrt(2*((local_del*invsig[row] + 2*subseqlen*mu[row]*mu[row]*invsig[row]*invsig[row]) + 2*subseqlen) - DUL2[row]) - DUL[row]);

    if (lb > bsf) {
        cnt++;
        lb_vector [row] = true;
    }
    else
    {
        col = start_pos;
        row = start_pos + diag - 1;
        tt = (cov_U_plus_cov_L_sec - mu[col] * sumU_sumL[row]) * invsig[col];
        local_del = norm_U_plus_norm_L_trans[row] - 2 * tt;
        lb = 0.5*(sqrt(2*((local_del + 2*subseqlen*mu[row]*mu[row]*invsig[row])*invsig[row] + 2*subseqlen) - DUL2[row]) - DUL[row]);

        if (lb>bsf) {
            lb_vector [col] = true;
            cnt++;
        }
    }

    for (int low_index = start_pos + 1; low_index < end_pos; low_index++) {

        int high_index = diag + low_index - 1;

        cov_U_plus_cov_L_fir = cov_U_plus_cov_L_fir -
                               dr_bwdU_plus_dr_bwdL[low_index] * dc_bwd[high_index] + dr_fwdU_plus_dr_fwdL[low_index] * dc_fwd[high_index];

        cov_U_plus_cov_L_sec = cov_U_plus_cov_L_sec -
                               dr_bwdU_plus_dr_bwdL[high_index] * dc_bwd[low_index] + dr_fwdU_plus_dr_fwdL[high_index] * dc_fwd[low_index];

        tt = (cov_U_plus_cov_L_fir - mu[high_index] * sumU_sumL[low_index]) * invsig[high_index];
        local_del = norm_U_plus_norm_L_trans[low_index] - 2 * tt;

        lb = 0.5*(sqrt(2*((local_del + 2*subseqlen*mu[low_index]*mu[low_index]*invsig[low_index])*invsig[low_index] + 2*subseqlen) - DUL2[low_index]) - DUL[low_index]);

        if (lb> bsf) {
            cnt++;
            lb_vector [low_index] = true;
            continue;
        }

        tt = (cov_U_plus_cov_L_sec - mu[low_index] * sumU_sumL[high_index]) * invsig[low_index];
        local_del = norm_U_plus_norm_L_trans[high_index] - 2 * tt;

        lb = 0.5*(sqrt(2*((local_del + 2*subseqlen*mu[high_index]*mu[high_index]*invsig[high_index])*invsig[high_index] + 2*subseqlen) - DUL2[high_index]) - DUL[high_index]);

        if (lb > bsf) {
            lb_vector [low_index] = true;
            cnt++;
        }

    }

}

