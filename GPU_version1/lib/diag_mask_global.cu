#include "hip/hip_runtime.h"

#include "matrix.cuh"
#include "iostream"
#include "vector"
#include "algorithm"
#include "cmath"
#include "chrono"
#include "GPU_parameters.h"
using namespace std;
#define BIAS 3

__device__ void
diag_mask_global(const FLOAT *ts, int subseqlen, int diagID, bool *lb_vector, const FLOAT *mu, const FLOAT *UTS,
                 const FLOAT *LTS, const FLOAT *MASK, const FLOAT *TS2, const FLOAT *sumMASK,
                 const FLOAT *invsig, const FLOAT *sumU_sumL, const FLOAT *dr_bwdU_plus_dr_bwdL,
                 const FLOAT *dr_fwdU_plus_dr_fwdL, const FLOAT *dc_bwd, const FLOAT *dc_fwd,
                 const FLOAT *dr_bwdMASK, const FLOAT *dr_fwdMASK, const FLOAT *dc_bwdTS2,
                 const FLOAT *dc_fwdTS2, const FLOAT *DUL2, FLOAT *lb_vector_new, FLOAT  bsf, FLOAT **subs,
                 FLOAT **special_shared_vector, FLOAT &cnt, const FLOAT *DUL,
                 const FLOAT *norm_U_plus_norm_L_global, int start_pos, int end_pos)
{
    FLOAT  M_NORM,t,dist,DUL_value;

    int diag = diagID;
    FLOAT  cov_U_plus_cov_L_fir = (elementWiseMultiply_p_plus_sum(ts + start_pos + diag - 1 + 3,
                                                                  UTS + start_pos + 3,LTS + start_pos + 3,subseqlen - 6));
    FLOAT  dot_TS_M_fir = (elementWiseMultiply_p_sum(ts + start_pos  + diag - 1 + 3,
                                                     MASK + start_pos + 3,subseqlen - 6));
    FLOAT  dot_TS2_M_fir = (elementWiseMultiply_p_sum(TS2 + start_pos + diag - 1 + 3,
                                                      MASK + start_pos + 3,subseqlen - 6));
    FLOAT  cov_U_plus_cov_L_sec = (elementWiseMultiply_p_plus_sum(ts + start_pos + 3 , UTS+ start_pos  + diag - 1 + 3,
                                                                  LTS + start_pos  + diag - 1 + 3,subseqlen - 6));
    FLOAT  dot_TS_M_sec = (elementWiseMultiply_p_sum(ts + start_pos + 3,
                                                     MASK + start_pos  + diag - 1 + 3,subseqlen - 6));
    FLOAT  dot_TS2_M_sec = (elementWiseMultiply_p_sum(TS2 + start_pos + 3,
                                                      MASK + start_pos + diag - 1 + 3,subseqlen - 6));

    int row;
    int col;
    FLOAT  lb;
    if(!lb_vector[0])
    {
        row = start_pos;
        col = start_pos + diag - 1;

        M_NORM = dot_TS2_M_fir - 2 * mu[col] * dot_TS_M_fir + sumMASK[row] * mu[col] * mu[col];
        M_NORM = M_NORM * invsig[col] * invsig[col];

        t = (cov_U_plus_cov_L_fir - mu[col] * sumU_sumL[row] +
             2 * mu[row] * (sumMASK[row] * mu[col] - dot_TS_M_fir)) * (invsig[col] * invsig[row]);
        dist = 2 * M_NORM - 2 * t;

        if(dist < DUL2[row]){
            lb = 0;
        }
        else{
            lb = 0.5*(sqrt(2*dist - DUL2[row]) - DUL[row]);
        }
        lb_vector_new [0] = lb;
        if(lb > bsf){
            lb_vector[0] = true;
            cnt++;
        }
        else
        {
            col = start_pos;
            row = start_pos + diag - 1;

            M_NORM = dot_TS2_M_sec - 2 * mu[col] * dot_TS_M_sec + sumMASK[row] * mu[col] * mu[col];
            M_NORM = M_NORM * invsig[col] * invsig[col];

            t = (cov_U_plus_cov_L_sec - mu[col] * sumU_sumL[row] +
                 2 * mu[row] * (sumMASK[row] * mu[col] - dot_TS_M_sec)) * (invsig[col] * invsig[row]);
            dist = 2 * M_NORM - 2 * t;

            if(dist < DUL2[row]){
                lb = 0;
            }
            else {
                lb = 0.5 * (sqrt(2 * dist - DUL2[row]) - DUL[row]);
            }
            lb_vector_new [0] = lb;
            if(lb > bsf)
            {
                lb_vector[0] = true;
                cnt++;
            }
        }
    }

    int lb_pos = 0;
    for (int low_index = start_pos + 1; low_index < end_pos; low_index++) {
        int high_index = diag + low_index - 1;
        lb_pos++;

        cov_U_plus_cov_L_fir = cov_U_plus_cov_L_fir - dr_bwdU_plus_dr_bwdL[low_index ] * dc_bwd[high_index ] +
                               (dr_fwdU_plus_dr_fwdL[low_index ]) * dc_fwd[high_index];
        dot_TS_M_fir = dot_TS_M_fir - dr_bwdMASK[low_index] * dc_bwd[high_index] + dr_fwdMASK[low_index] * dc_fwd[high_index];
        dot_TS2_M_fir = dot_TS2_M_fir - dr_bwdMASK[low_index] * dc_bwdTS2[high_index] + dr_fwdMASK[low_index] * dc_fwdTS2[high_index];

        cov_U_plus_cov_L_sec = cov_U_plus_cov_L_sec - dr_bwdU_plus_dr_bwdL[high_index] * dc_bwd[low_index] +
                               (dr_fwdU_plus_dr_fwdL[high_index]) * dc_fwd[low_index];
        dot_TS_M_sec = dot_TS_M_sec - dr_bwdMASK[high_index] * dc_bwd[low_index] + dr_fwdMASK[high_index] * dc_fwd[low_index];
        dot_TS2_M_sec = dot_TS2_M_sec - dr_bwdMASK[high_index] * dc_bwdTS2[low_index] + dr_fwdMASK[high_index] * dc_fwdTS2[low_index];

        if(lb_vector[lb_pos])
        {
            continue;
        }

        M_NORM = dot_TS2_M_fir - 2 * mu[high_index] * dot_TS_M_fir + sumMASK[low_index] * mu[high_index] * mu[high_index];
        M_NORM = M_NORM * invsig[high_index] * invsig[high_index];
        t = (cov_U_plus_cov_L_fir - mu[high_index] * sumU_sumL[low_index] +
             2 * mu[low_index] * (sumMASK[low_index] * mu[high_index] - dot_TS_M_fir)) * (invsig[high_index] * invsig[low_index]);
        dist = 2 * M_NORM - 2 * t  + norm_U_plus_norm_L_global[low_index];

        if(dist < DUL2[low_index])
        {
            lb = 0;
        }
        else
        {

            lb = 0.5*(sqrt(2 * dist - DUL2[low_index]) - DUL[low_index]) ;

            lb_vector_new [lb_pos] = lb;
            if(threadIdx.x == 0)

            if(lb > bsf)
            {

                lb_vector[lb_pos] = true;

                cnt++;
                continue;
            }
        }

        M_NORM = dot_TS2_M_sec - 2 * mu[low_index] * dot_TS_M_sec + sumMASK[high_index] * mu[low_index] * mu[low_index];
        M_NORM = M_NORM * invsig[low_index] * invsig[low_index];
        t = (cov_U_plus_cov_L_sec - mu[low_index] * sumU_sumL[high_index] +
             2 * mu[high_index] * (sumMASK[high_index] * mu[low_index] - dot_TS_M_sec)) * (invsig[low_index] * invsig[high_index]);
        dist = 2 * M_NORM - 2 * t + norm_U_plus_norm_L_global[high_index] ;

        if(dist < DUL2[high_index])
        {
            lb = 0;
        }
        else {
            lb = 0.5 * (sqrt(2 * dist - DUL2[high_index]) - DUL[high_index]);
            lb_vector_new [lb_pos] = max(lb, lb_vector_new [lb_pos]);

            if(lb > bsf)
            {
                lb_vector[lb_pos] = true;
                cnt++;

            }
        }

    }

    lb_pos = -1;
    auto temp_1 = end_pos;
    for (col = start_pos; col < temp_1; col++)
    {
        lb_pos++;
        row = diag + col - 1;

        if(lb_vector[lb_pos])
        {
            continue;
        }

        const FLOAT *t_ = subs[row];
        const FLOAT *q = subs[col];

        int m = subseqlen;
        FLOAT d;
        FLOAT threshold = bsf;
        FLOAT threshold2= threshold*threshold;

        FLOAT x0 = t_[0] ;
        FLOAT y0 = t_[(m - 1 )] ;

        const FLOAT dleft_orgin=DIST(x0, q[0]);
        const FLOAT dright_orgin=DIST(y0, q[m - 1]);
        FLOAT dleft = dleft_orgin;
        FLOAT dright = dright_orgin;

        FLOAT x1 = (t_[( 1)] );
        const FLOAT d_left_weak = min(DIST(x1, q[0]), DIST(x1, q[1]));
        d = min(d_left_weak, DIST(x0, q[1]));
        dleft+=d;

        FLOAT y1 = (t_[(m - 2 )]);
        const FLOAT d_right_weak = min(DIST(y1, q[m - 1]),  DIST(y1, q[m - 2]));
        d = min(d_right_weak,DIST(y0, q[m - 2]));
        dright+=d;

        if (dleft+dright + lb_vector_new [lb_pos] * lb_vector_new [lb_pos]>=threshold2){
            cnt++;
            lb_vector[lb_pos] = true;
            continue;
        }
        else{

            d = MIN(DIST(x1,q[0]) + DIST(t_[2],q[0]),d_left_weak + DIST(t_[2],q[1]));
            d = MIN(d,DIST(q[1],t_[1]) + DIST(q[2],t_[2]));
            d = MIN(d,dleft + DIST(q[2],t_[1]));
            d = MIN(d,DIST(q[1],t_[0]) + DIST(q[2],t_[0]));
            dleft = d + dleft_orgin;

            d = MIN(DIST(t_[m-2],q[m-1]) + DIST(t_[m-3],q[m-1]),d_right_weak + DIST(t_[m-3],q[m-2]));
            d = MIN(d,DIST(q[m-2],t_[m-2]) + DIST(q[m-3],t_[m-3]));
            d = MIN(d,dright + DIST(q[m-3],t_[m-2]));
            d = MIN(d,DIST(q[m-2],t_[m-1]) + DIST(q[m-3],t_[m-1]));
            dright = d + dright_orgin;

            if (dleft+dright  + lb_vector_new [lb_pos]* lb_vector_new [lb_pos] >=threshold2){
                lb_vector[lb_pos] = true;
                cnt++;
                continue;
            }

        }

    }

}

