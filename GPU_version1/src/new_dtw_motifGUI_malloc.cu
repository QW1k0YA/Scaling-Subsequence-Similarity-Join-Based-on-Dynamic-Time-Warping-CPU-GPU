#include "hip/hip_runtime.h"

#include <cstring>
#include <iomanip>
#include "../alldef/matrix.cuh"
#include "iostream"
#include "vector"
#include "algorithm"
#include "cmath"
#include "chrono"
#include "../alldef/allstruct.cuh"
#include "../alldef/typedefdouble.cuh"
#include "../allunder/underdtw.cuh"
#include "warp_num_per_block.h"
#include "collect_indices_kernal_after_keogh.h"
#include "process_keogh_and_dtw_kernal.cuh"
#include "device_prefix_sum.cuh"
#include <hip/hip_runtime_api.h>
#define DOUBLE_BIAS 6
#define BIAS 3

using namespace  std;
__device__ hipStream_t stream_pool[STREAM_POOL_SIZE_DTW];
__global__ void init_stream_pool(hipStream_t* streams) {
    int tid = threadIdx.x;
    if (tid < STREAM_POOL_SIZE_DTW) {
        hipStreamCreateWithFlags(&streams[tid], hipStreamNonBlocking);
    }
}

void new_dtw_motifGUI_malloc(const vector<FLOAT> &a, int subseqLen, int maxwarp, const vector<FLOAT> &mp_ed,
                             const vector<int> &mp_ed_index, const char *file)
{
    int subcount = a.size() - subseqLen + 1;
    int len = a.size();

    hipSetDevice(0);
    int device;
    hipGetDevice(&device);
    hipDeviceProp_t prop;
    hipError_t errO = hipGetDeviceProperties(&prop, device);
    int sm_count = prop.multiProcessorCount;

    cout << "sm = " << sm_count << endl;
    
    int THREADS_NUM = GRID_SIZE*BLOCK_SIZE;

    vector<FLOAT > mu(subcount),sig(subcount);
    mvmean(a,subseqLen,mu,sig);
    vector<FLOAT > invsig(subcount);
    vector<FLOAT > invsig_2(subcount);
    for(int i = 0;i < subcount;i ++)
    {
        invsig[i] = 1/sig[i];
        invsig_2[i] = invsig[i]*invsig[i];
    }

    FLOAT ** my_subs = (FLOAT **)malloc(subcount * sizeof(FLOAT *));
    FLOAT ** subs_U = (FLOAT **)malloc(subcount * sizeof(FLOAT *));
    FLOAT ** subs_L = (FLOAT **)malloc(subcount * sizeof(FLOAT *));

    for (int i = 0; i < subcount; ++i) {
        
        my_subs[i] = (FLOAT *)calloc(subseqLen, sizeof(FLOAT ));  
        subs_U[i] = (FLOAT *)calloc(subseqLen, sizeof(FLOAT ));
        subs_L[i] = (FLOAT *)calloc(subseqLen, sizeof(FLOAT ));
    }
    size_t temp_1;
    for(int i = 0;i < subcount;i ++)
    {
        for(int j = 0;j < subseqLen;j++)
        {
            my_subs[i][j] = (a[i + j] - mu[i]) * invsig[i];
        }
        lower_upper_lemire(my_subs[i],subseqLen,maxwarp,subs_L[i],subs_U[i]);
    }

    auto my_total_time = std::chrono::high_resolution_clock::now();
    printf("buffer_length: %d, maxwarp: %d\n",subseqLen,maxwarp);

    int minspacing = subseqLen;

    int k = 100;
    vector<int> min_index = min_v_k(mp_ed,k);

    FLOAT best_so_far = mp_ed[min_index[0]];
    std::vector<FLOAT> cb_temp(subseqLen + maxwarp + 1, 0);

    int low_index;
    int high_index;
    for(int i = 0;i < k;i++)
    {
        low_index = min_index[i];
        high_index = mp_ed_index[min_index[i]] - 1;

        if(LB_KK_FIRST(my_subs[low_index], my_subs[high_index], subs_U[high_index], subs_L[low_index], subseqLen,
                       best_so_far * best_so_far, cb_temp))
        {
            FLOAT dtw_value = sqrt(MON_dtw_host(my_subs[low_index], my_subs[high_index], cb_temp.data(),
                                                 subseqLen, maxwarp, best_so_far * best_so_far));

            if(dtw_value < best_so_far)
            {
                best_so_far = dtw_value;
            }
        }

    }

    cout << "best so far is " << best_so_far << endl;
    cout << "the buffer_length of TS is " <<  a.size() << endl;
    printf("################Start calculating lower bound#################  \n");

    int debug_sum = 0;
    FLOAT  bsf = best_so_far;

    bool * lb_vector= (bool*) malloc(subcount*sizeof(FLOAT ));
    
    subcount = a.size() - subseqLen + 1;

    int warpmax = maxwarp;

    vector<FLOAT > UTS(len);
    vector<FLOAT > LTS(len);
    lower_upper_lemire(a,len,warpmax,LTS,UTS);

    vector<FLOAT > UTS_p(subseqLen);
    vector<FLOAT > LTS_p(subseqLen);

    vector<FLOAT > dr_bwdU, dr_bwdL, dc_bwd;
    dr_bwdU = addElementToFront(extr_vfromv(UTS, 1, subcount - 1), 0.0);
    dr_bwdL = addElementToFront(extr_vfromv(LTS, 1, subcount - 1), 0.0);
    dc_bwd = addElementToFront(extr_vfromv(a, 1, subcount - 1), 0.0);

    vector<FLOAT > dr_fwdU, dr_fwdL, dc_fwd;
    dr_fwdU = extr_vfromv(UTS, subseqLen, len);
    dr_fwdL = extr_vfromv(LTS, subseqLen, len);
    dc_fwd = extr_vfromv(a, subseqLen, len);

    vector<FLOAT > dr_bwdU_plus_dr_bwdL, dr_fwdU_plus_dr_fwdL;
    dr_bwdU_plus_dr_bwdL = plusvector(dr_bwdU, dr_bwdL);
    dr_fwdU_plus_dr_fwdL = plusvector(dr_fwdU, dr_fwdL);

    const  vector<FLOAT >sumU = movsum(UTS, subseqLen - 1);
    const  vector<FLOAT >sumL = movsum(LTS, subseqLen - 1);
    const  vector<FLOAT >sumU2 = movsum(elementWiseMultiply(UTS, UTS), subseqLen - 1);
    const  vector<FLOAT >sumL2 = movsum(elementWiseMultiply(LTS, LTS), subseqLen - 1);
    const  vector<FLOAT >sumU2_sumL2 = plusvector(sumU2, sumL2);
    const  vector <FLOAT> sumU_sumL = plusvector(sumU, sumL);

    vector<FLOAT > del(subcount, 0);

    vector<FLOAT > normLTS(subseqLen, 0.0), normUTS(subseqLen, 0.0);
    vector<FLOAT > DUL2_fast(subcount);
    vector<FLOAT > DUL_fast(subcount);

    for (int row = 0; row < subcount; row++) {
        for (int i = row; i <= row + subseqLen - 1; i++) {
            normLTS[i - row] = (LTS[i] - mu[row])*invsig[row];
            normLTS[i - row] = (LTS[i] - mu[row])*invsig[row];
            normUTS[i - row] = (UTS[i] - mu[row])*invsig[row];
        }

        DUL2_fast[row] = pow(norm_vector(normLTS, 2), 2) + pow(norm_vector(normUTS, 2), 2) -
                         2 * sum_vector((elementWiseMultiply(normLTS, normUTS)));
        DUL_fast[row] = pow(MAX(DUL2_fast[row], 0.0), 0.5);

    }

    vector<FLOAT > norm_U_plus_norm_L_trans(subcount, 0.0);

    for (int row = 0; row < subcount; row++) {
        norm_U_plus_norm_L_trans[row] = (sumU2_sumL2[row] - 2 * sumU_sumL[row] * mu[row]) * invsig[row];
    }

    int len_of_table_local=INITIAL_LEN_OF_TABLE;

    FLOAT  real_max,real_min;
    vector<FLOAT > pos_UU(len, 1.0);
    vector<FLOAT > pos_LL(len, 1.0);
    int len_of_cdf;
     real_max = my_subs[0][0];
    real_min = my_subs[0][0];

    for(int row = 0;row < subcount;row++){
        for(int col = 0;col < subseqLen;col++){
            if(my_subs[row][col]> real_max && my_subs[row][col] < 100000) real_max = my_subs[row][col];
            if(my_subs[row][col] < real_min  && my_subs[row][col] > -100000) real_min = my_subs[row][col];
        }
    }

    FLOAT MAX_REAL_VALUE = MAX(abs(real_max), abs(real_min));
    if(MAX_REAL_VALUE > 3){
        len_of_cdf = (MAX_REAL_VALUE - 3)*20+600;
    }
    else{len_of_cdf = 600;
    }

    FLOAT  cov_U_plus_cov_L;

    FLOAT  pruning_rate = 0;
    FLOAT  thr = 0.9;

    int minlag = subseqLen;

    vector<FLOAT > proj(subseqLen);
    FLOAT lbk1,lbk2;

    vector<FLOAT > row_min_DEL(subcount, INFINITY);

    FLOAT  max_real_value = floor(MAX(abs(real_max), abs(real_min))) * 2 * 100;
    auto len_of_table_global = static_cast<long long>(MAX(INITIAL_LEN_OF_TABLE, max_real_value));

    vector<vector<FLOAT >> count_table_global(subseqLen, vector<FLOAT >(len_of_table_global, 0.0));

    printf("LEN_OF_TABLE_GLOBAL is %lld \n", len_of_table_global);
    compute_shared_data_global(a, subseqLen,my_subs,len_of_table_global,count_table_global);

    vector<FLOAT > MASK_global(len, 0.0);
    FLOAT  qqq,www;
    vector<FLOAT > TS2 = elementWiseMultiply(a, a);
    compute_MASK_global(a, subseqLen, len_of_table_global, count_table_global, subs_U, subs_L, MASK_global);

    int i;
    vector<FLOAT > UTS_global = elementWiseMultiply(UTS, MASK_global);
    vector<FLOAT > LTS_global = elementWiseMultiply(LTS, MASK_global);

    vector<FLOAT > dr_bwdU_global = addElementToFront(extr_vfromv(UTS_global, 1 + BIAS, UTS.size() - 1), 0.0);
    vector<FLOAT > dr_bwdL_global = addElementToFront(extr_vfromv(LTS_global, 1 + BIAS, LTS.size() - 1), 0.0);

    vector<FLOAT > dr_fwdU_global = extr_vfromv(UTS_global, subseqLen - BIAS, UTS_global.size());
    vector<FLOAT > dr_fwdL_global = extr_vfromv(LTS_global, subseqLen - BIAS, LTS_global.size());

    vector<FLOAT >dc_bwd_global = addElementToFront(extr_vfromv(a, 1 + BIAS, len), 0.0);
    vector<FLOAT >dc_fwd_global = extr_vfromv(a, subseqLen - BIAS, len);

    vector<FLOAT > dr_bwdU_plus_dr_bwdL_global, dr_fwdU_plus_dr_fwdL_global;
    dr_bwdU_plus_dr_bwdL_global = plusvector(dr_bwdU_global, dr_bwdL_global);
    dr_fwdU_plus_dr_fwdL_global = plusvector(dr_fwdU_global, dr_fwdL_global);

    const vector<FLOAT >sumU_global = movsum_p(UTS_global.data() + BIAS, len - DOUBLE_BIAS , subseqLen - 1 - DOUBLE_BIAS);
    const vector<FLOAT >sumL_global = movsum_p(LTS_global.data() + BIAS, len - DOUBLE_BIAS, subseqLen - 1 - DOUBLE_BIAS);
    const vector<FLOAT >sumU2_global = movsum(elementWiseMultiply_p(UTS_global.data() + BIAS, UTS_global.data() + BIAS, len - DOUBLE_BIAS), subseqLen - 1 - DOUBLE_BIAS);
    const vector<FLOAT >sumL2_global = movsum(elementWiseMultiply_p(LTS_global.data() + BIAS, LTS_global.data() + BIAS, len - DOUBLE_BIAS), subseqLen - 1 - DOUBLE_BIAS);
    const vector<FLOAT >sumU2_sumL2_global = plusvector(sumU2_global, sumL2_global);
    const vector<FLOAT >sumU_sumL_global = plusvector(sumU_global, sumL_global);

    const vector<FLOAT > sumMASK_global = movsum_p(MASK_global.data() + BIAS, MASK_global.size() - DOUBLE_BIAS, subseqLen - 1 - DOUBLE_BIAS);
    vector<FLOAT > norm_U_plus_norm_L_global(subcount, 0.0);

    const vector<FLOAT > TS2_global = elementWiseMultiply(a, a);

    for(int row = 1;row <= subcount;row++)
    {
        norm_U_plus_norm_L_global[row - 1] = (sumU2_sumL2_global[row - 1] - 2 * sumU_sumL_global[row - 1] * mu[row - 1]
                                             + 2*sumMASK_global[row-1]*mu[row-1]*mu[row-1]) * invsig[row-1] * invsig[row-1];
    }

    vector<FLOAT > del_global(subcount, 0);
    vector<FLOAT > normLTS_global(subseqLen, 0.0), normUTS_global(subseqLen, 0.0);
    FLOAT  del_ths_global;
    vector<FLOAT  >DUL2_global(subcount);
    vector<FLOAT  >DUL_global(subcount);

    for(int row = 0;row < subcount;row++)
    {
        for (i = row; i <= row + subseqLen - 1; i++) {

            normLTS_global[i - row] = (LTS_global[i] - mu[row]) *invsig[row];
            normUTS_global[i - row] = (UTS_global[i] - mu[row]) *invsig[row];
        }

        DUL2_global[row] =pow(norm_vector(substractvector(normLTS_global,normUTS_global),2),2);
        
        DUL_global[row] = sqrt(MAX(DUL2_global[row],0));

    }

    const vector<FLOAT > dr_bwdMASK_global = addElementToFront(extr_vfromv(MASK_global, 1 + BIAS, MASK_global.size()), 0.0);
    const vector<FLOAT > dr_fwdMASK_global = extr_vfromv(MASK_global, subseqLen - BIAS,
                                                        MASK_global.size());

    const vector<FLOAT > dc_bwdTS2_global = addElementToFront(extr_vfromv(TS2, 1 + BIAS , TS2.size() - 1), 0.0);
    const vector<FLOAT > dc_fwdTS2_global = extr_vfromv(TS2, subseqLen - BIAS, TS2.size());

    vector<FLOAT > lb_vector_new(subcount);
    FLOAT **special_shared_vector = (FLOAT **)malloc(subcount * sizeof(FLOAT *));
    
    for (i = 0; i < subcount; i++) {
        special_shared_vector[i] = (FLOAT *)malloc(2 * sizeof(FLOAT));
        if (special_shared_vector[i] == nullptr) {

            for (int j = 0; j < i; j++) {
                free(special_shared_vector[j]);
            }
            free(special_shared_vector);
        }
    }
    for(i = 0;i < subcount;i++)
    {
        for(int j = 0;j < 2;j++)
        {
            special_shared_vector[i][j] = 0;
        }
    }

    int forsize_divide_100 = (subcount- minlag)/100 ;

    int i_ = 0;
    FLOAT fast_prune_cnt = 0;
    FLOAT local_prune_cnt = 0;
    FLOAT global_prune_cnt = 0;
    FLOAT KK_prunes_cnt = 0;
    FLOAT P_prunes_cnt = 0;
    FLOAT Dtw_prunes_cnt = 0;
    long long all_cnt = 0;

    vector<FLOAT > del_local(subcount, 0.0);
    vector<FLOAT > raw_DIFF_UL_local(UTS.size(), 0.0);
    vector<FLOAT > raw_DIFF_UL2_local (UTS.size(), 0.0);
    vector<FLOAT > raw_DIFF_UL2_temp_local (UTS.size(), 0.0);
    vector<FLOAT > DUL2_raw_local(subcount, 0.0);
    vector<FLOAT > DUL2_local(subcount, 0.0);

    vector<FLOAT > DUL_local(subcount);
    vector<FLOAT >sumU_local (subcount, 0.0);
    vector<FLOAT >sumL_local (subcount, 0.0);
    vector<FLOAT >sumU2_local  (subcount, 0.0);
    vector<FLOAT >sumL2_local (subcount, 0.0);
    vector<FLOAT >sumU2_sumL2_loacl (subcount, 0.0);
    vector<FLOAT >sumU_sumL_loacl (subcount, 0.0);

    vector<FLOAT > sumMASK_local(subcount, 0.0);
    vector<FLOAT > norm_U_plus_norm_L_local(subcount, 0.0);

    FLOAT  fast_time = 0;
    FLOAT  global_time = 0;
    FLOAT  local_up_time = 0;
    FLOAT  local_down_time = 0;

    int index_diag = 0;
    int alpha_interval = MAX(subseqLen*5,1000);
    int buffer_length = MAX(subseqLen/10,30);
    int buffer_div_alpha = ceil(1.0*alpha_interval/buffer_length);
    alpha_interval = buffer_div_alpha*buffer_length;

    int for_cnt = 0;
    FLOAT alpha = 0;
    FLOAT start_pos = -1.5;
    FLOAT end_pos = 1.5;
    vector<FLOAT > f_x;
    vector<FLOAT > X;
    vector<FLOAT > bsf_X;
    vector<bool> lb_vector_for_next(subcount);

    vector<FLOAT> alpha_table(subcount,0.0);
    FLOAT best_alpha = 0;

    FLOAT  *d_a, *d_TS2,*d_mu, *d_sig, *d_sumU_sumL, *d_invsig, *d_norm_U_plus_norm_L_trans, *d_del;
    FLOAT  *d_dr_bwdU_plus_dr_bwdL, *d_dc_bwd, *d_dr_fwdU_plus_dr_fwdL, *d_dc_fwd;
    FLOAT  *d_UTS, *d_LTS, *d_UTS_global, *d_LTS_global, *d_MASK_global;
    FLOAT  *d_pos_UU, *d_pos_LL;
    SHORT *d_count_table_local;
    
    FLOAT *d_sumMASK_global, *d_sumU_sumL_global;
    FLOAT *d_dr_bwdU_plus_dr_bwdL_global, *d_dr_fwdU_plus_dr_fwdL_global;
    FLOAT *d_dc_bwd_global, *d_dc_fwd_global;
    FLOAT *d_dr_bwdMASK_global, *d_dr_fwdMASK_global;
    FLOAT *d_dc_bwdTS2_global, *d_dc_fwdTS2_global;
    FLOAT *d_DUL2_global, *d_norm_U_plus_norm_L_global, *d_DUL_global;
    FLOAT *d_DUL_fast,*d_DUL2_fast;
    FLOAT  *d_bsf;
    FLOAT *d_cnt_up, *d_cnt_down;

    int *d_prefix_sum;
    hipMalloc(&d_prefix_sum, GRID_SIZE * sizeof(int)); CUERR
    int *d_num_inclusive;
    hipMalloc(&d_num_inclusive, THREADS_NUM * sizeof(int)); CUERR

    int init_d_num_of_dtw = 0;

    size_t *h_num_of_dtw;
    h_num_of_dtw = (size_t*) malloc(sizeof(size_t));
    h_num_of_dtw[0] = 0;

    hipMalloc(&d_a, a.size() * sizeof(FLOAT )); CUERR
    hipMemcpy(d_a, a.data(), a.size() * sizeof(FLOAT ), hipMemcpyHostToDevice);

    hipMalloc(&d_TS2, TS2.size() * sizeof(FLOAT )); CUERR
    hipMemcpy(d_TS2, TS2.data(), TS2.size() * sizeof(FLOAT ), hipMemcpyHostToDevice);

    hipMalloc(&d_mu, mu.size() * sizeof(FLOAT )); CUERR
    hipMemcpy(d_mu, mu.data(), mu.size() * sizeof(FLOAT ), hipMemcpyHostToDevice);

    hipMalloc(&d_sig, sig.size() * sizeof(FLOAT )); CUERR
    hipMemcpy(d_sig, sig.data(), sig.size() * sizeof(FLOAT ), hipMemcpyHostToDevice);

    hipMalloc(&d_sumU_sumL, sumU_sumL.size() * sizeof(FLOAT )); CUERR
    hipMemcpy(d_sumU_sumL, sumU_sumL.data(), sumU_sumL.size() * sizeof(FLOAT ), hipMemcpyHostToDevice);

    hipMalloc(&d_invsig, invsig.size() * sizeof(FLOAT )); CUERR
    hipMemcpy(d_invsig, invsig.data(), invsig.size() * sizeof(FLOAT ), hipMemcpyHostToDevice);

    hipMalloc(&d_norm_U_plus_norm_L_trans, norm_U_plus_norm_L_trans.size() * sizeof(FLOAT )); CUERR
    hipMemcpy(d_norm_U_plus_norm_L_trans, norm_U_plus_norm_L_trans.data(),
               norm_U_plus_norm_L_trans.size() * sizeof(FLOAT ), hipMemcpyHostToDevice);

    hipMalloc(&d_dr_bwdU_plus_dr_bwdL, dr_bwdU_plus_dr_bwdL.size() * sizeof(FLOAT )); CUERR
    hipMemcpy(d_dr_bwdU_plus_dr_bwdL, dr_bwdU_plus_dr_bwdL.data(),
               dr_bwdU_plus_dr_bwdL.size() * sizeof(FLOAT ), hipMemcpyHostToDevice);

    hipMalloc(&d_dc_bwd, dc_bwd.size() * sizeof(FLOAT )); CUERR
    hipMemcpy(d_dc_bwd, dc_bwd.data(), dc_bwd.size() * sizeof(FLOAT ), hipMemcpyHostToDevice);

    hipMalloc(&d_dr_fwdU_plus_dr_fwdL, dr_fwdU_plus_dr_fwdL.size() * sizeof(FLOAT )); CUERR
    hipMemcpy(d_dr_fwdU_plus_dr_fwdL, dr_fwdU_plus_dr_fwdL.data(),
               dr_fwdU_plus_dr_fwdL.size() * sizeof(FLOAT ), hipMemcpyHostToDevice);

    hipMalloc(&d_dc_fwd, dc_fwd.size() * sizeof(FLOAT )); CUERR
    hipMemcpy(d_dc_fwd, dc_fwd.data(), dc_fwd.size() * sizeof(FLOAT ), hipMemcpyHostToDevice);

    hipMalloc(&d_UTS, UTS.size() * sizeof(FLOAT )); CUERR
    hipMemcpy(d_UTS, UTS.data(), UTS.size() * sizeof(FLOAT ), hipMemcpyHostToDevice);

    hipMalloc(&d_LTS, LTS.size() * sizeof(FLOAT )); CUERR
    hipMemcpy(d_LTS, LTS.data(), LTS.size() * sizeof(FLOAT ), hipMemcpyHostToDevice);

    hipMalloc(&d_UTS_global, UTS_global.size() * sizeof(FLOAT )); CUERR
    hipMemcpy(d_UTS_global, UTS_global.data(), UTS_global.size() * sizeof(FLOAT ), hipMemcpyHostToDevice);

    hipMalloc(&d_LTS_global, LTS_global.size() * sizeof(FLOAT )); CUERR
    hipMemcpy(d_LTS_global, LTS_global.data(), LTS_global.size() * sizeof(FLOAT ), hipMemcpyHostToDevice);

    hipMalloc(&d_MASK_global, MASK_global.size() * sizeof(FLOAT )); CUERR
    hipMemcpy(d_MASK_global, MASK_global.data(), MASK_global.size() * sizeof(FLOAT ), hipMemcpyHostToDevice);

    hipMalloc(&d_pos_UU, pos_UU.size() * sizeof(FLOAT )); CUERR
    hipMemcpy(d_pos_UU, pos_UU.data(), pos_UU.size() * sizeof(FLOAT ), hipMemcpyHostToDevice);

    hipMalloc(&d_pos_LL, pos_LL.size() * sizeof(FLOAT )); CUERR
    hipMemcpy(d_pos_LL, pos_LL.data(), pos_LL.size() * sizeof(FLOAT ), hipMemcpyHostToDevice);

    hipMalloc(&d_sumMASK_global, sumMASK_global.size() * sizeof(FLOAT)); CUERR
    hipMemcpy(d_sumMASK_global, sumMASK_global.data(),
               sumMASK_global.size() * sizeof(FLOAT), hipMemcpyHostToDevice);

    hipMalloc(&d_sumU_sumL_global, sumU_sumL_global.size() * sizeof(FLOAT)); CUERR
    hipMemcpy(d_sumU_sumL_global, sumU_sumL_global.data(),
               sumU_sumL_global.size() * sizeof(FLOAT), hipMemcpyHostToDevice);

    hipMalloc(&d_dr_bwdU_plus_dr_bwdL_global, dr_bwdU_plus_dr_bwdL_global.size() * sizeof(FLOAT)); CUERR
    hipMemcpy(d_dr_bwdU_plus_dr_bwdL_global, dr_bwdU_plus_dr_bwdL_global.data(),
               dr_bwdU_plus_dr_bwdL_global.size() * sizeof(FLOAT), hipMemcpyHostToDevice);

    hipMalloc(&d_dr_fwdU_plus_dr_fwdL_global, dr_fwdU_plus_dr_fwdL_global.size() * sizeof(FLOAT)); CUERR
    hipMemcpy(d_dr_fwdU_plus_dr_fwdL_global, dr_fwdU_plus_dr_fwdL_global.data(),
               dr_fwdU_plus_dr_fwdL_global.size() * sizeof(FLOAT), hipMemcpyHostToDevice);

    hipMalloc(&d_dc_bwd_global, dc_bwd_global.size() * sizeof(FLOAT)); CUERR
    hipMemcpy(d_dc_bwd_global, dc_bwd_global.data(),
               dc_bwd_global.size() * sizeof(FLOAT), hipMemcpyHostToDevice);

    hipMalloc(&d_dc_fwd_global, dc_fwd_global.size() * sizeof(FLOAT)); CUERR
    hipMemcpy(d_dc_fwd_global, dc_fwd_global.data(),
               dc_fwd_global.size() * sizeof(FLOAT), hipMemcpyHostToDevice);

    hipMalloc(&d_dr_bwdMASK_global, dr_bwdMASK_global.size() * sizeof(FLOAT)); CUERR
    hipMemcpy(d_dr_bwdMASK_global, dr_bwdMASK_global.data(),
               dr_bwdMASK_global.size() * sizeof(FLOAT), hipMemcpyHostToDevice);

    hipMalloc(&d_dr_fwdMASK_global, dr_fwdMASK_global.size() * sizeof(FLOAT)); CUERR
    hipMemcpy(d_dr_fwdMASK_global, dr_fwdMASK_global.data(),
               dr_fwdMASK_global.size() * sizeof(FLOAT), hipMemcpyHostToDevice);

    hipMalloc(&d_dc_bwdTS2_global, dc_bwdTS2_global.size() * sizeof(FLOAT)); CUERR
    hipMemcpy(d_dc_bwdTS2_global, dc_bwdTS2_global.data(),
               dc_bwdTS2_global.size() * sizeof(FLOAT), hipMemcpyHostToDevice);

    hipMalloc(&d_dc_fwdTS2_global, dc_fwdTS2_global.size() * sizeof(FLOAT)); CUERR
    hipMemcpy(d_dc_fwdTS2_global, dc_fwdTS2_global.data(),
               dc_fwdTS2_global.size() * sizeof(FLOAT), hipMemcpyHostToDevice);

    hipMalloc(&d_DUL2_global, DUL2_global.size() * sizeof(FLOAT)); CUERR
    hipMemcpy(d_DUL2_global, DUL2_global.data(),
               DUL2_global.size() * sizeof(FLOAT), hipMemcpyHostToDevice);

    hipMalloc(&d_norm_U_plus_norm_L_global, norm_U_plus_norm_L_global.size() * sizeof(FLOAT)); CUERR
    hipMemcpy(d_norm_U_plus_norm_L_global, norm_U_plus_norm_L_global.data(),
               norm_U_plus_norm_L_global.size() * sizeof(FLOAT), hipMemcpyHostToDevice);

    hipMalloc(&d_DUL_global, DUL_global.size() * sizeof(FLOAT)); CUERR
    hipMemcpy(d_DUL_global, DUL_global.data(),
               DUL_global.size() * sizeof(FLOAT), hipMemcpyHostToDevice);

    hipMalloc(&d_DUL_fast, DUL_fast.size() * sizeof(FLOAT)); CUERR
    hipMemcpy(d_DUL_fast, DUL_fast.data(),
               DUL_fast.size() * sizeof(FLOAT), hipMemcpyHostToDevice);
    hipMalloc(&d_DUL2_fast, DUL2_fast.size() * sizeof(FLOAT)); CUERR
    hipMemcpy(d_DUL2_fast, DUL2_fast.data(),
               DUL2_fast.size() * sizeof(FLOAT), hipMemcpyHostToDevice);

    int num_threads = THREADS_NUM;

    bool* d_lb_vector;
    hipMalloc(&d_lb_vector, num_threads * STEP_LENGTH * sizeof(bool));
    bool* h_lb_vector = new bool[num_threads * STEP_LENGTH];
    memset(h_lb_vector,0,sizeof(bool)*num_threads * STEP_LENGTH);
    hipMemcpy(d_lb_vector, h_lb_vector, num_threads * STEP_LENGTH * sizeof(bool), hipMemcpyHostToDevice);

    int* d_indices;
    hipMalloc(&d_indices, num_threads * STEP_LENGTH * sizeof(int)); CUERR
    int* d_diag;
    hipMalloc(&d_diag, num_threads * STEP_LENGTH * sizeof(int)); CUERR

    int* h_indices = new int[num_threads * STEP_LENGTH];
    memset( h_indices,0,sizeof(int)*num_threads * STEP_LENGTH);
    hipMemcpy(d_diag,  h_indices, num_threads * STEP_LENGTH * sizeof(int), hipMemcpyHostToDevice);
    int* h_indices2 = new int[num_threads * STEP_LENGTH];
    memset( h_indices2,0,sizeof(int)*num_threads * STEP_LENGTH);
    hipMemcpy(d_indices,  h_indices, num_threads * STEP_LENGTH * sizeof(int), hipMemcpyHostToDevice);

    FLOAT*  d_lb_vector_new;
    hipMalloc(&d_lb_vector_new, num_threads * STEP_LENGTH * sizeof(FLOAT));

    cout << "best so far is " << bsf << endl;
    FLOAT* h_bsf = (FLOAT *)malloc(BSF_POOL * sizeof(FLOAT));
    for(int i = 0; i < BSF_POOL; i++)
    {
        h_bsf[i] = bsf;
    }
    FLOAT*  d_bsf_global = nullptr;

    hipMalloc(&d_bsf_global, BSF_POOL * sizeof(FLOAT));
    hipMemcpy(d_bsf_global, h_bsf, BSF_POOL * sizeof(FLOAT), hipMemcpyHostToDevice);

    FLOAT ** d_my_subs = nullptr;
    FLOAT ** d_subs_U = nullptr;
    FLOAT ** d_subs_L = nullptr;
    FLOAT ** d_shared_special_vector = nullptr;

    hipMalloc(&d_my_subs, subcount * sizeof(FLOAT *)); CUERR
    hipMalloc(&d_subs_U, subcount * sizeof(FLOAT *)); CUERR
    hipMalloc(&d_subs_L, subcount * sizeof(FLOAT *)); CUERR
    hipMalloc(&d_shared_special_vector, subcount *sizeof(FLOAT *)); CUERR

    FLOAT ** tmp_my_subs = (FLOAT **)malloc(subcount * sizeof(FLOAT *));
    FLOAT ** tmp_subs_U = (FLOAT **)malloc(subcount * sizeof(FLOAT *));
    FLOAT ** tmp_subs_L = (FLOAT **)malloc(subcount * sizeof(FLOAT *));
    FLOAT ** tmp_d_shared_special_vector = (FLOAT **)malloc(subcount * sizeof(FLOAT *));

    for (int i = 0; i < subcount; ++i) {
        
        hipMalloc(&tmp_my_subs[i], subseqLen * sizeof(FLOAT )); CUERR
        hipMalloc(&tmp_subs_U[i], subseqLen * sizeof(FLOAT )); CUERR
        hipMalloc(&tmp_subs_L[i], subseqLen * sizeof(FLOAT )); CUERR
        hipMalloc(&tmp_d_shared_special_vector[i],  2 * sizeof(FLOAT )); CUERR

        hipMemcpy(tmp_my_subs[i], my_subs[i],
                   subseqLen * sizeof(FLOAT ), hipMemcpyHostToDevice);
        hipMemcpy(tmp_subs_U[i], subs_U[i],
                   subseqLen * sizeof(FLOAT ), hipMemcpyHostToDevice);
        hipMemcpy(tmp_subs_L[i], subs_L[i],
                   subseqLen * sizeof(FLOAT ), hipMemcpyHostToDevice);
        hipMemcpy(tmp_d_shared_special_vector[i], special_shared_vector[i],
                   2 * sizeof(FLOAT ), hipMemcpyHostToDevice);
    }

    hipMemcpy(d_my_subs, tmp_my_subs,
               subcount * sizeof(FLOAT *), hipMemcpyHostToDevice);
    hipMemcpy(d_subs_U, tmp_subs_U,
               subcount * sizeof(FLOAT *), hipMemcpyHostToDevice);
    hipMemcpy(d_subs_L, tmp_subs_L,
               subcount * sizeof(FLOAT *), hipMemcpyHostToDevice);
    hipMemcpy(d_shared_special_vector, tmp_d_shared_special_vector,
               2 * sizeof(FLOAT *), hipMemcpyHostToDevice);

    for (int i = 0; i < subcount; ++i) {
        free(my_subs[i]);
        free(subs_U[i]);
        free(subs_L[i]);
    }

    const int block_size = BLOCK_SIZE;       
    const int num_blocks = GRID_SIZE; 

    hipError_t error = hipDeviceSetLimit(hipLimitPrintfFifoSize, 1024 * 1024);
    if (error != hipSuccess) {

    }

    int step = STEP_LENGTH;

    if (errO != hipSuccess) {
        std::cerr << "Failed to get device properties: " << hipGetErrorString(errO) << std::endl;
    }
    int numBlocks;
    int blockSize = 32*WARP_NUMS;

    hipMemcpy(&bsf, d_bsf_global, sizeof(FLOAT), hipMemcpyDeviceToHost);

    cout << "initial bsf = " << bsf << endl;

    FLOAT t_first = clock();
    double t_total_dtw = 0;
    double t_total_diag = 0;
    int d_start_pos,d_end_pos;
    int t_for_cnt = ceil(1.0*subcount/step);

    size_t dtw_cnt = 0;
    int DTW_SHARED_MEM_SIZE = 2*subseqLen*sizeof(FLOAT)*WARP_NUMS;

    bool profileflag = true;

    int bl_size = ceil(maxwarp/31.0);
    if(bl_size == 2)
    {
        bl_size = 3;
    }
    else if(bl_size == 4)
    {
        bl_size = 5;
    }
    for(int t_index = 0;t_index < t_for_cnt;t_index ++)
    {
        d_start_pos = t_index * step;
        d_end_pos = d_start_pos + step;

        for (int diag = minlag + 1; diag <= subcount; diag += THREADS_NUM)
        {
            if (d_start_pos > subcount - diag + 1) {
                continue;
            }

            FLOAT t1 = clock();

            GLOBAL_DIAG<<<num_blocks, block_size>>>(
                    minlag,
                    subcount,
                    subseqLen,
                    len,
                    warpmax,
                    d_a,          
                    d_mu,
                    d_sumU_sumL,
                    d_invsig,
                    d_norm_U_plus_norm_L_trans,
                    d_dr_bwdU_plus_dr_bwdL,
                    d_dc_bwd,
                    d_dr_fwdU_plus_dr_fwdL,
                    d_dc_fwd,
                    d_UTS,
                    d_LTS,
                    d_UTS_global,
                    d_LTS_global,
                    d_MASK_global,
                    d_TS2,
                    d_lb_vector,
                    d_lb_vector_new,
                    d_sumMASK_global,
                    d_sumU_sumL_global,
                    d_dr_bwdU_plus_dr_bwdL_global,
                    d_dr_fwdU_plus_dr_fwdL_global,
                    d_dc_bwd_global,
                    d_dc_fwd_global,
                    d_dr_bwdMASK_global,
                    d_dr_fwdMASK_global,
                    d_dc_bwdTS2_global,
                    d_dc_fwdTS2_global,
                    d_DUL2_global,
                    d_DUL_global,
                    d_norm_U_plus_norm_L_global,
                    d_my_subs,
                    d_shared_special_vector,
                    d_bsf_global, d_DUL_fast,
                    d_DUL2_fast, diag, d_start_pos, d_end_pos);

            hipError_t syncErr = hipDeviceSynchronize();
            if (syncErr != hipSuccess) {
                printf("fail in diag %s\n", hipGetErrorString(syncErr));
                printf("diag = %d st = %d end = %d\n",diag,d_start_pos,d_end_pos);
            }

            hipMemcpy(d_prefix_sum, &init_d_num_of_dtw, sizeof(int), hipMemcpyHostToDevice);

            FLOAT t2 = clock();
            
            calculate_num_for_each_block<<<GRID_SIZE, BLOCK_SIZE>>>(d_lb_vector, subcount,
                                                                    d_start_pos, d_end_pos,
                                                                    diag, d_prefix_sum, d_num_inclusive);
            hipDeviceSynchronize();

            device_prefix_sum_block<<<1, GRID_SIZE>>>(d_prefix_sum);

            hipDeviceSynchronize();

            collect_indices<<<GRID_SIZE, BLOCK_SIZE>>>(d_diag, d_indices, d_lb_vector, subcount,
                                                       d_start_pos, d_end_pos,
                                                       diag, d_prefix_sum, d_num_inclusive);

            hipMemcpy(h_num_of_dtw, &d_prefix_sum[GRID_SIZE - 1], sizeof(int), hipMemcpyDeviceToHost);
            
            dtw_cnt+= h_num_of_dtw[0];
            syncErr = hipDeviceSynchronize();
            if (syncErr != hipSuccess) {
                printf("fail in collect %s\n", hipGetErrorString(syncErr));
            }

            FLOAT t3 = clock();

            if(subseqLen < 1024)
            {

                process_keogh_and_dtw_kernel_for_a_Parallelogram<<<h_num_of_dtw[0], 32, DTW_SHARED_MEM_SIZE>>>
                        (d_my_subs, d_subs_L, d_subs_U, subseqLen, d_bsf_global, subcount,
                         warpmax, d_indices, d_diag);

            }
            else
            {

                process_keogh_and_dtw_kernel_for_a_Parallelogram_without_shared_memory_and_nomalized<<<h_num_of_dtw[0], 32>>>
                        (d_my_subs, d_UTS, d_LTS, d_mu, d_invsig, subseqLen, d_bsf_global, subcount,
                         warpmax, d_indices, d_diag, bl_size);

            }
            syncErr = hipDeviceSynchronize();
            if (syncErr != hipSuccess) {
                printf("fail in process %s\n", hipGetErrorString(syncErr));
                printf("diag = %d st = %d end = %d\n",diag,d_start_pos,d_end_pos);
            }

            syn_bsfs<<<1,BSF_POOL>>>(d_bsf_global);

            hipMemcpy(&bsf, d_bsf_global, sizeof(FLOAT), hipMemcpyDeviceToHost);

            FLOAT t4 = clock();
            t_total_dtw += (t4 - t3) ;
            t_total_diag += (t2 - t1) ;

        }
    }

    hipMemcpy(&bsf, d_bsf_global, sizeof(FLOAT), hipMemcpyDeviceToHost);

    printf("dtw cnt = %llu\n",dtw_cnt);

    FLOAT t_last = clock();
    printf("dtw time : %fs,diag time : %fs \n",t_total_dtw/CLOCKS_PER_SEC,t_total_diag/CLOCKS_PER_SEC);
    printf("total time is %fs   input of bsf:%5.3f  ,final distance :%f  \n", FLOAT(t_last - t_first)/CLOCKS_PER_SEC, best_so_far, bsf);

    FILE *op = fopen(file,"a");
    fprintf(op, "%lf",  FLOAT(t_last - t_first)/CLOCKS_PER_SEC);
    fprintf(op, "\n");
    fclose(op);

    for (int i = 0; i < subcount; ++i) {
        hipFree(tmp_my_subs[i]);
        hipFree(tmp_subs_U[i]);
        hipFree(tmp_subs_L[i]);
    }
    hipFree(d_my_subs);
    hipFree(d_subs_U);
    hipFree(d_subs_L);
    hipFree(d_bsf_global);

    free(tmp_my_subs);
    free(tmp_subs_U);
    free(tmp_subs_L);

    hipFree(d_a);
    hipFree(d_TS2);
    hipFree(d_mu);
    hipFree(d_sig);
    hipFree(d_sumU_sumL);
    hipFree(d_invsig);
    hipFree(d_norm_U_plus_norm_L_trans);

    hipFree(d_dr_bwdU_plus_dr_bwdL);
    hipFree(d_dc_bwd);
    hipFree(d_dr_fwdU_plus_dr_fwdL);
    hipFree(d_dc_fwd);
    hipFree(d_UTS);
    hipFree(d_LTS);
    hipFree(d_UTS_global);
    hipFree(d_LTS_global);
    hipFree(d_MASK_global);
    hipFree(d_pos_UU);
    hipFree(d_pos_LL);
    hipFree(d_DUL2_global);
    hipFree(d_DUL_global);
    hipFree(d_DUL2_fast);
    hipFree(d_DUL_fast);

    hipFree(d_lb_vector);
    hipFree(d_lb_vector_new);
    hipFree(d_indices);
    hipFree(d_diag);
    hipFree(d_prefix_sum);
    hipFree(d_num_inclusive);

    cout << endl;

    free(h_num_of_dtw);
    free(lb_vector);

    for (int j = 0; j < subcount; j++) {
        free(special_shared_vector[j]);
    }
    free(special_shared_vector);

    free(my_subs);
    free(subs_U);
    free(subs_L);

    delete[] h_lb_vector;
    delete[] h_indices;
}

