#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include "GPU_parameters.h"
#include "matrix.cuh"
#include "warp_num_per_block.h"

#define THREAD_NUM_PER_WARP 32

__global__ void process_dtw_kernel
        (FLOAT **my_subs, int *d_indices, int subseqLen, int diag_offset, FLOAT *bsf, int subcount, int diag,
         int *d_counter, int ran_idx, int w, FLOAT *cb1, FLOAT *cb2)
{
    
    int *indices_local = &d_indices[subcount*(diag_offset - diag)];
    extern __shared__ FLOAT shared_mem[];
    
    FLOAT* q = &shared_mem[0];                  
    FLOAT* t = &shared_mem[subseqLen*WARP_NUMS];

    FLOAT threshold = bsf[ran_idx];
    FLOAT threshold_2 = threshold * threshold;
    int bid = blockIdx.x;

    for (int i = bid; i < d_counter[diag_offset - diag]; i += gridDim.x*WARP_NUMS ) {
        int i_block = threadIdx.x/32;
        int original_index = indices_local[i*WARP_NUMS+i_block];

        {
            FLOAT*  sub1 = my_subs[original_index];
            FLOAT*  sub2 = my_subs[original_index + diag_offset - 1];

            if(original_index + diag_offset - 1 >= subcount){
                printf("dtw_kernal 67 %d %d\n",original_index ,diag_offset - 1);
            }
            FLOAT result = 0.0f;

            int bl_size;
            bl_size= ceil(w/31.0);

            if(w < 32)
            {

            }
            else{
                DTW_stairs_for_block(sub1, sub2, result, subseqLen, threshold_2, w, q, t,bl_size);
            }

            __syncthreads();

            if(threadIdx.x%32 == 16)
            {

                if(result < threshold)
                {
                    atomicMinFloat(&bsf[ran_idx], result);
                    __threadfence();
                }
            }

        }

    }
}

__global__ void process_keogh_and_dtw_kernel_for_a_Parallelogram
        (FLOAT **my_subs, FLOAT **my_L, FLOAT **my_U, int subseqLen, FLOAT *bsf,
         int subcount, int w, const int *indices, const int *diag_of_indices,
         int num_of_dtw)
{
    extern __shared__ FLOAT shared_mem[];
    
    FLOAT* q = &shared_mem[0];                  
    FLOAT* t = &shared_mem[subseqLen*WARP_NUMS];

    FLOAT cb1;
    FLOAT cb2;
    
    int bid = blockIdx.x;
    int tid = threadIdx.x%32;
    int step = (subseqLen/32);
    FLOAT threshold = bsf[bid%BSF_POOL]; 
    FLOAT threshold_2 = threshold * threshold;

    int i = bid;

    {
        int original_index = indices[i];
        int diag_offset = diag_of_indices[i];

        if(diag_offset != 0) 
        {

            FLOAT*  sub1 = my_subs[original_index];
            FLOAT*  sub2 = my_subs[original_index + diag_offset - 1];

            FLOAT* U1 = my_U[original_index];
            FLOAT* L1 = my_L[original_index];
            FLOAT* U2 = my_U[original_index + diag_offset - 1];
            FLOAT* L2 = my_L[original_index + diag_offset - 1];

            FLOAT partial_dist1 = 0;
        partial_dist1 = lb_keogh_warp(sub1+ tid *step, U2 + tid*step, L2+ tid*step, step);
            FLOAT partial_dist2 = 0;
        partial_dist2 = lb_keogh_warp(sub2+ tid*step, U1+ tid*step, L1+ tid*step, step);

#pragma unroll
            for (int mask = 1; mask <= 16; mask <<= 1) {
                
                FLOAT tmp1 = __shfl_down_sync(0xFFFFFFFF, partial_dist1, mask);
                FLOAT tmp2 = __shfl_down_sync(0xFFFFFFFF, partial_dist2, mask);
                if (tid + mask < 32) {
                    partial_dist1 += tmp1;
                    partial_dist2 += tmp2;
                }
            }
            cb1 = partial_dist1;
            cb2 = partial_dist2;

            bool flag = 1;
            if (tid == 0 && (partial_dist1 > threshold_2 || partial_dist2 > threshold_2)) {
                
                flag = false; 

            }
            flag = __shfl_sync(0x1F, flag, 0);
            __syncwarp();

            if(flag)
            {

                FLOAT result = 0.0f;

                int bl_size;
                bl_size= ceil(w/31.0);

                if(w < 32)
                {
                    DTW_stairs(sub1, sub2, result, subseqLen, threshold_2, w, q, t, cb1, cb2);
                }
                else{
                    DTW_stairs_for_block(sub1, sub2, result, subseqLen, threshold_2, w, q, t,bl_size);
                }

                __syncwarp();
                if(threadIdx.x%32 == 16)
                {
                    if(result < threshold)
                    {

                        atomicMinFloat(&bsf[bid%BSF_POOL], result);

                        __threadfence();
                    }
                }
            }
        }

    }

}

__global__ void process_keogh_and_dtw_kernel_for_a_Parallelogram_without_shared_memory
        (FLOAT **my_subs, FLOAT **my_L, FLOAT **my_U, int subseqLen, FLOAT *bsf,
         int subcount, int w, const int *indices, const int *diag_of_indices,
         int num_of_dtw)
{
    extern __shared__ FLOAT shared_mem[];
    
    FLOAT* q = &shared_mem[0];                  
    FLOAT* t = &shared_mem[subseqLen*WARP_NUMS];

    FLOAT cb1;
    FLOAT cb2;
    
    int bid = blockIdx.x;
    int tid = threadIdx.x%32;
    int step = (subseqLen/32);
    FLOAT threshold = bsf[bid%BSF_POOL]; 
    FLOAT threshold_2 = threshold * threshold;

    int i = bid;

    {
        int original_index = indices[i];
        int diag_offset = diag_of_indices[i];

        if(diag_offset != 0) 
        {

            FLOAT*  sub1 = my_subs[original_index];
            FLOAT*  sub2 = my_subs[original_index + diag_offset - 1];

            FLOAT* U1 = my_U[original_index];
            FLOAT* L1 = my_L[original_index];
            FLOAT* U2 = my_U[original_index + diag_offset - 1];
            FLOAT* L2 = my_L[original_index + diag_offset - 1];

            FLOAT partial_dist1 = 0;
            partial_dist1 = lb_keogh_warp(sub1+ tid *step, U2 + tid*step, L2+ tid*step, step);
            FLOAT partial_dist2 = 0;
            partial_dist2 = lb_keogh_warp(sub2+ tid*step, U1+ tid*step, L1+ tid*step, step);

#pragma unroll
            for (int mask = 1; mask <= 16; mask <<= 1) {
                
                FLOAT tmp1 = __shfl_down_sync(0xFFFFFFFF, partial_dist1, mask);
                FLOAT tmp2 = __shfl_down_sync(0xFFFFFFFF, partial_dist2, mask);
                if (tid + mask < 32) {
                    partial_dist1 += tmp1;
                    partial_dist2 += tmp2;
                }
            }
            cb1 = partial_dist1;
            cb2 = partial_dist2;

            bool flag = 1;
            if (tid == 0 && (partial_dist1 > threshold_2 || partial_dist2 > threshold_2)) {
                
                flag = false; 

            }
            flag = __shfl_sync(0x1F, flag, 0);
            __syncwarp();

            if(flag)
            {

                FLOAT result = 0.0f;

                int bl_size;
                bl_size= ceil(w/31.0);

                if(w < 32)
                {
                    DTW_stairs_without_shared(sub1, sub2, result, subseqLen, threshold_2, w, q, t);
                }
                else{
                    DTW_stairs_for_block_without_shared(sub1, sub2, result, subseqLen, threshold_2, w, q, t,bl_size);
                }

                __syncwarp();
                if(threadIdx.x%32 == 16)
                {

                    if(result < threshold)
                    {

                        atomicMinFloat(&bsf[bid%BSF_POOL], result);

                        __threadfence();
                    }
                }
            }
        }

    }

}

__global__ void process_keogh_and_dtw_kernel_for_a_Parallelogram_keogh_prune
        (FLOAT **my_subs, FLOAT **my_L, FLOAT **my_U, int subseqLen, FLOAT *bsf,
         int subcount, int w, const int *indices, const int *diag_of_indices,
         int *num_prune)
{
    extern __shared__ FLOAT shared_mem[];
    
    FLOAT* q = &shared_mem[0];                  
    FLOAT* t = &shared_mem[subseqLen*WARP_NUMS];

    FLOAT cb1;
    FLOAT cb2;
    
    int bid = blockIdx.x;
    int tid = threadIdx.x%32;
    int step = (subseqLen/32);
    FLOAT threshold = bsf[bid%BSF_POOL]; 
    FLOAT threshold_2 = threshold * threshold;

    int i = bid;

    {
        int original_index = indices[i];
        int diag_offset = diag_of_indices[i];

        if(diag_offset != 0) 
        {

            FLOAT*  sub1 = my_subs[original_index];
            FLOAT*  sub2 = my_subs[original_index + diag_offset - 1];

            FLOAT* U1 = my_U[original_index];
            FLOAT* L1 = my_L[original_index];
            FLOAT* U2 = my_U[original_index + diag_offset - 1];
            FLOAT* L2 = my_L[original_index + diag_offset - 1];

            FLOAT partial_dist1 = 0;
            partial_dist1 = lb_keogh_warp(sub1+ tid *step, U2 + tid*step, L2+ tid*step, step);
            FLOAT partial_dist2 = 0;
            partial_dist2 = lb_keogh_warp(sub2+ tid*step, U1+ tid*step, L1+ tid*step, step);

#pragma unroll
            for (int mask = 1; mask <= 16; mask <<= 1) {
                
                FLOAT tmp1 = __shfl_down_sync(0xFFFFFFFF, partial_dist1, mask);
                FLOAT tmp2 = __shfl_down_sync(0xFFFFFFFF, partial_dist2, mask);
                if (tid + mask < 32) {
                    partial_dist1 += tmp1;
                    partial_dist2 += tmp2;
                }
            }

            cb1 = partial_dist1;
            cb2 = partial_dist2;

            bool flag = 1;
            if (tid == 0 && (partial_dist1 > threshold_2 || partial_dist2 > threshold_2)) {

                flag = false; 

            }

            flag = __shfl_sync(0x1F, flag, 0);

            __syncwarp();
            if(flag)
            {

                FLOAT result = 0.0f;

                int bl_size;
                bl_size= ceil(w/31.0);

                if(w < 32)
                {

                    DTW_stairs(sub1, sub2, result, subseqLen, threshold_2, w, q, t, cb1, cb2);
                }
                else{
                    DTW_stairs_for_block(sub1, sub2, result, subseqLen, threshold_2, w, q, t,bl_size);
                }

                __syncwarp();
                if(threadIdx.x%32 == 16)
                {
                    if(result < threshold)
                    {
                        atomicMinFloat(&bsf[bid%BSF_POOL], result);

                        __threadfence();
                    }
                }
            }
        }

    }

}

__global__ void process_MON_dtw_kernel
        (FLOAT **my_subs, int *d_indices, int subseqLen, int diag_offset, FLOAT *bsf, int subcount,
         int diag, int *d_counter, int w, int ran_idx)
{
    
    extern __shared__ FLOAT mon_buffers[];
    
    int *indices_local = &d_indices[subcount*(diag_offset - diag)];

    FLOAT threshold = bsf[ran_idx];
    FLOAT threshold_2 = threshold * threshold;
    int tid = blockIdx.x*blockDim.x + threadIdx.x;
    int stride = blockDim.x*gridDim.x;

    FLOAT *local_buffer = &mon_buffers[threadIdx.x*(2*subseqLen + 2)];

    for (int i = tid; i < d_counter[diag_offset - diag]; i += stride ) {

        int original_index = indices_local[i];

        indices_local[i] = 0;
        
        FLOAT*  sub1 = my_subs[original_index];
        FLOAT*  sub2 = my_subs[original_index + diag_offset - 1];

        if(original_index + diag_offset - 1 >= subcount){
            printf("dtw_kernal 67 %d %d\n",original_index ,diag_offset - 1);
        }
        
        FLOAT result = 0.0f;
        result = sqrt(MON_dtw(sub1, sub2, subseqLen, w, threshold_2, local_buffer,tid));
        if(result < 5)
        {
            printf("oi !!!%d %d %f\n",original_index,original_index + diag_offset - 1,result);
        }

        if(threadIdx.x == blockDim.x - 1)
        {
            if(diag_offset == 6544 && original_index == 2396)
            {
               printf("dtw_value = %f \n" ,result);
            }

            if(result < threshold)
            {
                atomicMinFloat(&bsf[ran_idx], result);
                __threadfence();
            }
        }

    }
}