#include "hip/hip_runtime.h"

#include "../alldef/matrix.cuh"
#include "iostream"
#include "vector"
#include "algorithm"
#include "cmath"
#include "chrono"

using namespace std;
void
LB_KIM(FLOAT threshold, int m, FLOAT **special_shared_vector, const vector<vector<FLOAT >> &subs, int temp_1,
       int diag, vector<bool> &lb_vector, FLOAT &cnt) {

    for (int col = 1; col <= temp_1; col++) {

        int row = diag + col - 1;

        if(lb_vector[col - 1])
        {
            
            continue;
        }
        const vector<FLOAT > &t_ = subs[row - 1];
        const vector<FLOAT > &q = subs[col - 1];

        FLOAT  d;
        FLOAT threshold2=threshold*threshold;

        FLOAT x0 = t_[0] ;
        FLOAT y0 = t_[(m - 1 )] ;

        const FLOAT dleft_orgin=DIST(x0, q[0]);
        const FLOAT dright_orgin=DIST(y0, q[m - 1]);
        FLOAT dleft = dleft_orgin;
        FLOAT dright = dright_orgin;

        FLOAT x1 = (t_[( 1)] );
        const FLOAT d_left_weak = min(DIST(x1, q[0]), DIST(x1, q[1]));
        d = min(d_left_weak, DIST(x0, q[1]));
        dleft+=d;

        FLOAT y1 = (t_[(m - 2 )]);
        const FLOAT d_right_weak = min(DIST(y1, q[m - 1]),  DIST(y1, q[m - 2]));
        d = min(d_right_weak,DIST(y0, q[m - 2]));
        dright+=d;

        if (dleft+dright>=threshold2){
            cnt++;
            lb_vector[col] = true;
            continue;
        }
        else{

            d = MIN(DIST(x1,q[0]) + DIST(t_[2],q[0]),d_left_weak + DIST(t_[2],q[1]));
            d = MIN(d,DIST(q[1],t_[1]) + DIST(q[2],t_[2]));
            d = MIN(d,dleft + DIST(q[2],t_[1]));
            d = MIN(d,DIST(q[1],t_[0]) + DIST(q[2],t_[0]));
            dleft = d + dleft_orgin;

            d = MIN(DIST(t_[m-2],q[m-1]) + DIST(t_[m-3],q[m-1]),d_right_weak + DIST(t_[m-3],q[m-2]));
            d = MIN(d,DIST(q[m-2],t_[m-2]) + DIST(q[m-3],t_[m-3]));
            d = MIN(d,dright + DIST(q[m-3],t_[m-2]));
            d = MIN(d,DIST(q[m-2],t_[m-1]) + DIST(q[m-3],t_[m-1]));
            dright = d + dright_orgin;

            if (dleft+dright >=threshold2){
                lb_vector[col] = true;
                cnt++;
                continue;
            }
            special_shared_vector[col][0]=dleft; special_shared_vector[col][1]=dright;
        }
    }

}

void LB_KIM_new(FLOAT threshold, int m, FLOAT **special_shared_vector, const vector<vector<FLOAT >> &subs, int temp_1,
                int diag, vector<bool> &lb_vector, FLOAT &cnt) {

    for (int col = 1; col <= temp_1; col++) {
        int row = diag + col - 1;

        if (lb_vector[col - 1]) {
            continue;
        }
        const vector<FLOAT > &t_ = subs[row - 1];
        const vector<FLOAT > &q = subs[col - 1];

        FLOAT d;
        FLOAT threshold2 = threshold * threshold;

        FLOAT x0 = t_[0];
        FLOAT y0 = t_[(m - 1)];

        const FLOAT dleft_orgin = DIST(x0, q[0]);
        const FLOAT dright_orgin = DIST(y0, q[m - 1]);
        FLOAT dleft = dleft_orgin;
        FLOAT dright = dright_orgin;

        FLOAT x1 = (t_[(1)]);
        const FLOAT d_left_weak = min(DIST(x1, q[0]), DIST(x1, q[1]));
        d = min(d_left_weak, DIST(x0, q[1]));
        dleft += d;

        FLOAT y1 = (t_[(m - 2)]);
        const FLOAT d_right_weak = min(DIST(y1, q[m - 1]), DIST(y1, q[m - 2]));
        d = min(d_right_weak, DIST(y0, q[m - 2]));
        dright += d;

        if (dleft + dright  >= threshold2) {
            cnt++;
            lb_vector[col] = true;
            continue;
        } else {

            d = MIN(DIST(x1, q[0]) + DIST(t_[2], q[0]), d_left_weak + DIST(t_[2], q[1]));
            d = MIN(d, DIST(q[1], t_[1]) + DIST(q[2], t_[2]));
            d = MIN(d, dleft + DIST(q[2], t_[1]));
            d = MIN(d, DIST(q[1], t_[0]) + DIST(q[2], t_[0]));
            dleft = d + dleft_orgin;

            d = MIN(DIST(t_[m - 2], q[m - 1]) + DIST(t_[m - 3], q[m - 1]), d_right_weak + DIST(t_[m - 3], q[m - 2]));
            d = MIN(d, DIST(q[m - 2], t_[m - 2]) + DIST(q[m - 3], t_[m - 3]));
            d = MIN(d, dright + DIST(q[m - 3], t_[m - 2]));
            d = MIN(d, DIST(q[m - 2], t_[m - 1]) + DIST(q[m - 3], t_[m - 1]));
            dright = d + dright_orgin;

            if (dleft + dright>= threshold2) {
                lb_vector[col] = true;
                cnt++;
                continue;
            }
            special_shared_vector[col][0] = dleft;
            special_shared_vector[col][1] = dright;
        }
    }
}
