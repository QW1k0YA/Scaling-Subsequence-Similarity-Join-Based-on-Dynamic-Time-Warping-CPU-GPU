#include "hip/hip_runtime.h"

#include "matrix.cuh"
#include "../alldef/collect_indices_kernal_after_keogh.h"
#include <hipcub/hipcub.hpp>
__global__ void
collect_indices_kernel_after_keogh(int *d_indices, bool *lb_vector, int subcount,
                                   int start_pos, int end_pos, int diag,
                                   int *d_counter) {
    int tid =blockIdx.x * blockDim.x + threadIdx.x;
    bool *lb_local = &lb_vector[tid*subcount];
    int* indices_local = &d_indices[tid*subcount];
    diag = diag + tid;

    if(start_pos > subcount - diag + 1) return;
    end_pos = MIN(end_pos, subcount - diag + 1);

    d_counter[tid] = 0;
    for(int i = start_pos;i < end_pos;i++)
    {
        if ( !lb_local[i]) {
            indices_local[d_counter[tid]++] = i;
        }
        else
        {
            lb_local[i] = false;
        }
    }
}

__global__ void
calculate_num_for_each_block(bool *lb_vector, int subcount, int start_pos, int end_pos, int diag, int *global_offset,
                             int *num_inclusive) {
    typedef hipcub::BlockScan<int, BLOCK_SIZE> BlockScan;
    __shared__ typename BlockScan::TempStorage temp_storage;

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    bool *lb_local = &lb_vector[tid * STEP_LENGTH];
    diag = diag + tid;

    end_pos = MIN(end_pos, subcount - diag + 1);

    int num_in_the_thread = 0;
    int lb_pos = 0;
    for (int i = start_pos; i < end_pos; i++) {

        if (!lb_local[lb_pos]) {
            num_in_the_thread++;
        }

        lb_pos++;
    }

    int init_value = 0;
    __syncthreads();
    BlockScan(temp_storage).InclusiveSum(num_in_the_thread, num_inclusive[tid],init_value);

    if (threadIdx.x == blockDim.x - 1) {
        global_offset[blockIdx.x] = num_inclusive[tid];
    }

}

__global__ void
collect_indices(int *d_diag, int *d_indices, bool *lb_vector, int subcount, int start_pos, int end_pos, int diag,
                const int *global_offset, const int *num_inclusive) {
    
    int tid =blockIdx.x * blockDim.x + threadIdx.x;
    bool *lb_local = &lb_vector[tid*STEP_LENGTH];
    int bid = blockIdx.x;

    int global_off = (bid > 0) ? global_offset[bid - 1] : 0;
    int block_off = (threadIdx.x >0) ? num_inclusive[tid - 1] : 0;
    diag = diag + tid;
    if(start_pos > subcount - diag + 1) return;
    end_pos = MIN(end_pos, subcount - diag + 1);

    int* indices_local = &d_indices[block_off + global_off];
    int* diag_local = &d_diag[block_off + global_off];
    unsigned int pos_for_indices = 0;
    unsigned int pos_for_diag_local = 0;

    int lb_pos = 0;
    for(int i = start_pos;i < end_pos;i++)
    {

        if ( !lb_local[lb_pos]) {
            indices_local[pos_for_indices++] = i;
            diag_local[pos_for_diag_local++] = diag;
        }
        else
        {
            lb_local[lb_pos] = false;
        }
        lb_pos++;
    }
}

