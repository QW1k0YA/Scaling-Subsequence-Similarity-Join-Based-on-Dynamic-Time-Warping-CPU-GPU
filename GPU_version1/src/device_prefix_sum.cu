#include "hip/hip_runtime.h"

#include "device_prefix_sum.cuh"
#include <hipcub/hipcub.hpp>
#include "GPU_parameters.h"
void device_prefix_sum(int *d_array, int n) {
    void *d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;

    hipcub::DeviceScan::InclusiveSum(d_temp_storage, temp_storage_bytes, d_array, d_array, n);
    hipMalloc(&d_temp_storage, temp_storage_bytes);
    hipcub::DeviceScan::InclusiveSum(d_temp_storage, temp_storage_bytes, d_array, d_array, n);
    hipFree(d_temp_storage);
}

__global__ void device_prefix_sum_block(int *d_out)
{
    
    typedef hipcub::BlockScan<int, GRID_SIZE> BlockScan;
    __shared__ typename BlockScan::TempStorage temp_storage;
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int thread_data = d_out[tid];
    BlockScan(temp_storage).InclusiveSum(thread_data, thread_data);
    __syncthreads();
    d_out[tid] = thread_data;

}