#include "hip/hip_runtime.h"

#include "../alldef/matrix.cuh"
#include "algorithm"
#include "cmath"
#include "chrono"
using namespace std;

int FIND_POS_LOCAL_f(FLOAT input,FLOAT max_abs_value,int length)
{
    int pos;
    if(input >  3)
    {
        pos = 300 + (input - 3)*10;
    }
    else if(input < -3)
    {
        pos =  - 300  + (input + 3)*10;
    }
    else
    {
        pos = input * 100;
    }

    pos+= length/2;
    return  pos;
}

void
compute_shared_data_local(const vector<FLOAT > &ts, int subseqlen, const vector<vector<FLOAT >> &subs,
                          const vector<vector<FLOAT >> &UU, const vector<vector<FLOAT >> &LL, FLOAT  &real_min,
                          FLOAT  &real_max, vector<FLOAT> &pos_UU, vector<FLOAT> &pos_LL, int &len_of_cdf,
                          vector<vector<short>> &count_table_cdf, FLOAT MAX_REAL_VALUE)
{

    int subcount = ts.size() - subseqlen + 1;
    auto start_time = std::chrono::high_resolution_clock::now();
    FLOAT  max_possible_value = floor(sqrt(subseqlen));

    vector<FLOAT > cnt(ts.size(), 0.0);
    vector<FLOAT > strange_count(ts.size(), 0.0);
    int pp;
    FLOAT  t;
    size_t ts_size = ts.size();
    for(int j = 0;j <subcount;j++)
    {

        for(int i = 0;i < subseqlen;i++)
        {
            
            FLOAT  value = subs[j][i];
            
            pp=FIND_POS_LOCAL_f(value,MAX_REAL_VALUE,len_of_cdf);
            
            if(pp < 0 )
            {
                pp = 0;
            }
            if(pp >= len_of_cdf)
            {
                pp = len_of_cdf - 1;
            }

            count_table_cdf[i+j][pp] = count_table_cdf[i+j][pp] + 1;
            
        }
    }

    for(int i = 0;i < ts_size;i++)
    {
        int sum1 = 0;
        for(int j = 0;j < len_of_cdf;j++)
        {
            sum1 = sum1 + count_table_cdf[i][j];
            count_table_cdf[i][j] = sum1;
        }
    }

    vector<FLOAT > sum_UU(ts.size(), 0.0);
    vector<FLOAT > sum_LL(ts.size(), 0.0);

    for(int i = 0;i < subcount;i++)
    {
        for(int j = 0;j < subseqlen;j++)
        {

            sum_UU[i+j] = sum_UU[i+j]+UU[i][j];
            sum_LL[i+j] = sum_LL[i+j]+LL[i][j];

            cnt[i+j] = cnt[i+j] + 1;
        }
    }
    sum_UU = elementWiseDivison_vv(sum_UU,cnt);
    sum_LL = elementWiseDivison_vv(sum_LL,cnt);

    FLOAT  posUU,posLL;

    for(int i = 0;i < ts_size;i++)
    {
        FLOAT  LL_temp = sum_LL[i];
        FLOAT  UU_temp = sum_UU[i];

        posLL = FIND_POS_LOCAL_f(LL_temp,MAX_REAL_VALUE,len_of_cdf);
        posUU = FIND_POS_LOCAL_f(UU_temp,MAX_REAL_VALUE,len_of_cdf);
        if(posLL < 0)
        {
            posLL = 1;
        }
        if(posLL > len_of_cdf)
        {
            posLL =  len_of_cdf;
        }
        if(posUU < 0)
        {
            posLL = 1;
        }
        if(posLL >  len_of_cdf )
        {
            posLL =  len_of_cdf;
        }
        if(posUU < 0)
        {
            posUU = 1;
        }
        if(posUU >  len_of_cdf )
        {
            posUU = len_of_cdf;
        }
        pos_UU[i] = posUU;
        pos_LL[i] = posLL;
    }
    printf("computing the Shared DATA: V1, LEN_OF_TABLE_LOCAL is %d \n", len_of_cdf);

}

void
compute_shared_data_local(const vector<FLOAT > &ts, int subseqlen, FLOAT **subs,
                          FLOAT **UU, FLOAT **LL, FLOAT  &real_min,
                          FLOAT  &real_max, vector<FLOAT> &pos_UU, vector<FLOAT> &pos_LL, int &len_of_cdf,
                          vector<vector<short>> &count_table_cdf, FLOAT MAX_REAL_VALUE)
{

    int subcount = ts.size() - subseqlen + 1;
    auto start_time = std::chrono::high_resolution_clock::now();
    FLOAT  max_possible_value = floor(sqrt(subseqlen));

    vector<FLOAT > cnt(ts.size(), 0.0);
    vector<FLOAT > strange_count(ts.size(), 0.0);
    int pp;
    FLOAT  t;
    size_t ts_size = ts.size();
    for(int j = 0;j <subcount;j++)
    {

        for(int i = 0;i < subseqlen;i++)
        {
            
            FLOAT  value = subs[j][i];
            
            pp=FIND_POS_LOCAL_f(value,MAX_REAL_VALUE,len_of_cdf);
            
            if(pp < 0 )
            {
                pp = 0;
            }
            if(pp >= len_of_cdf)
            {
                pp = len_of_cdf - 1;
            }

            count_table_cdf[i+j][pp] = count_table_cdf[i+j][pp] + 1;
            
        }
    }

    for(int i = 0;i < ts_size;i++)
    {
        int sum1 = 0;
        for(int j = 0;j < len_of_cdf;j++)
        {
            sum1 = sum1 + count_table_cdf[i][j];
            count_table_cdf[i][j] = sum1;
        }
    }

    vector<FLOAT > sum_UU(ts.size(), 0.0);
    vector<FLOAT > sum_LL(ts.size(), 0.0);

    for(int i = 0;i < subcount;i++)
    {
        for(int j = 0;j < subseqlen;j++)
        {

            sum_UU[i+j] = sum_UU[i+j]+UU[i][j];
            sum_LL[i+j] = sum_LL[i+j]+LL[i][j];

            cnt[i+j] = cnt[i+j] + 1;
        }
    }
    sum_UU = elementWiseDivison_vv(sum_UU,cnt);
    sum_LL = elementWiseDivison_vv(sum_LL,cnt);

    FLOAT  posUU,posLL;

    for(int i = 0;i < ts_size;i++)
    {
        FLOAT  LL_temp = sum_LL[i];
        FLOAT  UU_temp = sum_UU[i];

        posLL = FIND_POS_LOCAL_f(LL_temp,MAX_REAL_VALUE,len_of_cdf);
        posUU = FIND_POS_LOCAL_f(UU_temp,MAX_REAL_VALUE,len_of_cdf);
        if(posLL < 0)
        {
            posLL = 1;
        }
        if(posLL > len_of_cdf)
        {
            posLL =  len_of_cdf;
        }
        if(posUU < 0)
        {
            posLL = 1;
        }
        if(posLL >  len_of_cdf )
        {
            posLL =  len_of_cdf;
        }
        if(posUU < 0)
        {
            posUU = 1;
        }
        if(posUU >  len_of_cdf )
        {
            posUU = len_of_cdf;
        }
        pos_UU[i] = posUU;
        pos_LL[i] = posLL;
    }
    printf("computing the Shared DATA: V1, LEN_OF_TABLE_LOCAL is %d \n", len_of_cdf);

}

