#include "hip/hip_runtime.h"

#include "../alldef/matrix.cuh"
#include "iostream"
#include "vector"

#include "algorithm"
#include "cmath"
#include "chrono"
#include "../alldef/allstruct.cuh"
#include "fstream"
#include "../alldef/fileoperations.cuh"
#include "../alldef/matrix.cuh"
#include "../allunder/underdtw.cuh"

using namespace std;

int main(int argc,char* argv[])
{

    string filename1 = argv[1];
    const char* filename2 = argv[2];
    int m = atoi(argv[3]);
    float w = atof(argv[4]);

    vector<FLOAT > TS1 = loadfile(filename1);
    vector<FLOAT > TS_1;
    size_t num = TS1.size()/2;
    for(int i = 0;i < num/2  ;i++)
    {
        TS_1.push_back(TS1[i]);
    }

    vector<FLOAT> TS = TS1;
    DTWMotifDiscoveryGUI(TS,m,MAX(round(m*w),1),filename2);

}

