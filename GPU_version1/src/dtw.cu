#include "hip/hip_runtime.h"

#include "../alldef/matrix.cuh"
#include "iostream"
#include "vector"
#include "algorithm"
#include "cmath"
#include "chrono"
using namespace std;
#define INF 1E20
__device__ FLOAT dtw(const FLOAT *A, const FLOAT *B, int m, int r, FLOAT threshold_2,   FLOAT *cost, FLOAT *cost_prev)
{

    FLOAT *cost_tmp;
    int i,j,k;
    FLOAT x,y,z,min_cost;

    for(k=0; k<2*r+1; k++)    cost[k]=INF;

    for(k=0; k<2*r+1; k++)    cost_prev[k]=INF;

    for (i=0; i<m; i++)
    {
        k = MAX(0,r-i);
        
        min_cost = INF;
        int index;

        for(j=max(0,i-r); j<=MIN(m-1,i+r); j++, k++)
        {

            if ((i==0)&&(j==0))
            {
                cost[k]=DIST(A[0], B[0]);
                min_cost = cost[k];
                continue;
            }

            if ((j-1<0)||(k-1<0))     y = INF;
            else                      y = cost[k-1];
            if ((i-1<0)||(k+1>2*r))   x = INF;
            else                      x = cost_prev[k+1];
            if ((i-1<0)||(j-1<0))     z = INF;
            else                      z = cost_prev[k];

            cost[k] = MIN( MIN( x, y) , z) + DIST(A[i], B[j]);
            FLOAT i_ = i;
            FLOAT j_ = j;

            FLOAT  d_ = cost[k];
            if (cost[k] < min_cost)
            {
                min_cost = cost[k];
                
                index = k;
            }
        }

        cost_tmp = cost;
        cost = cost_prev;
        cost_prev = cost_tmp;
    }
    k--;

    FLOAT final_dtw = cost_prev[k];
    return sqrt(final_dtw);
}
FLOAT dtw(const vector<FLOAT> &A, const vector<FLOAT> &B, int m, int r, FLOAT threshold_2, vector<FLOAT> &cb)
{

    FLOAT *cost;
    FLOAT *cost_prev;
    FLOAT *cost_tmp;
    int i,j,k;
    FLOAT x,y,z,min_cost;

    cost = (FLOAT* )malloc(sizeof(double)*(2*r+1));
    for(k=0; k<2*r+1; k++)    cost[k]=INF;

    cost_prev = (FLOAT* )malloc(sizeof(double)*(2*r+1));
    for(k=0; k<2*r+1; k++)    cost_prev[k]=INF;

    for (i=0; i<m; i++)
    {
        k = MAX(0,r-i);
        
        min_cost = INF;
        int index;

        for(j=max(0,i-r); j<=MIN(m-1,i+r); j++, k++)
        {

            if ((i==0)&&(j==0))
            {
                cost[k]=DIST(A[0], B[0]);
                min_cost = cost[k];
                continue;
            }

            if ((j-1<0)||(k-1<0))     y = INF;
            else                      y = cost[k-1];
            if ((i-1<0)||(k+1>2*r))   x = INF;
            else                      x = cost_prev[k+1];
            if ((i-1<0)||(j-1<0))     z = INF;
            else                      z = cost_prev[k];

            cost[k] = MIN( MIN( x, y) , z) + DIST(A[i], B[j]);
            FLOAT i_ = i;
            FLOAT j_ = j;

            FLOAT  d_ = cost[k];
            if (cost[k] < min_cost)
            {
                min_cost = cost[k];
                
                index = k;
            }
        }

        if (i+r < m-1 && min_cost + cb[i+r+1] >= threshold_2)
        {   free(cost);
            free(cost_prev);
            return sqrt(min_cost + cb[i+r+1]);
        }

        cost_tmp = cost;
        cost = cost_prev;
        cost_prev = cost_tmp;
    }
    k--;

    FLOAT final_dtw = cost_prev[k];
    free(cost);
    free(cost_prev);
    return sqrt(final_dtw);
}

