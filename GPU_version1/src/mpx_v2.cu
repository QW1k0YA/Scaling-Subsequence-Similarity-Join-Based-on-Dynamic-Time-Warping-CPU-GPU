#include "hip/hip_runtime.h"

#include "../alldef/matrix.cuh"
#include "iostream"
#include "vector"
#include "algorithm"
#include "cmath"
#include "chrono"
#include "../alldef/allstruct.cuh"

#include "../allunder/undermpx_v2.cuh"

using namespace  std;

RETURN_MPX mpx_v2(const vector<FLOAT >& timeSeries, int minlag, int subseqlen){

    int subcount =timeSeries.size() - subseqlen + 1;

    vector<FLOAT > timeSeries_ = timeSeries;

    vector<FLOAT > nanmap = findNonZero(isinfinite(movsum(timeSeries_, subseqlen - 1)));
    vector<FLOAT > nanIDX = findNonZero(isNaN(timeSeries));

    for(auto value : nanIDX)
    {
        timeSeries_[value - 1] = 0;
    }

    vector<FLOAT > mu = moving_mean(timeSeries_, subseqlen);
    vector<FLOAT > mus = moving_mean(timeSeries_, subseqlen - 1);

    vector<FLOAT > invnorm(subcount, 0.0);

    vector<FLOAT > timeSeries_1 = timeSeries_;

    for(int i = 1;i <= subcount;i++)
    {
        vector<FLOAT > temp_ts2;
        for(int j = i;j <= i + subseqlen -1;j++)
        {
            temp_ts2.push_back(timeSeries_[j-1] - mu[i-1]);
        }

        invnorm[i-1] = 1/ norm_vector(temp_ts2,2);
    }

    for(auto value:nanmap)
    {
        invnorm[value-1] = NAN;
    }

    int j = 0;
    for(auto value: isinfinite(invnorm))
    {
        if(value == 1)
        {
            invnorm[j] = NAN;
        }
        j++;
    }

    vector<FLOAT > dr_bwd = addElementToFront(substractvector(extr_vfromv(timeSeries_, 1, subcount - 1),
                                                             extr_vfromv(mu, 1, subcount - 1)), 0);
    vector<FLOAT > dc_bwd = addElementToFront(substractvector(extr_vfromv(timeSeries_, 1, subcount - 1),
                                                             extr_vfromv(mus, 2, subcount)), 0);
    vector<FLOAT > dr_fwd = substractvector(extr_vfromv(timeSeries_, subseqlen, timeSeries_.size()),
                                           extr_vfromv(mu, 1, subcount));
    vector<FLOAT > dc_fwd = substractvector(extr_vfromv(timeSeries_, subseqlen, timeSeries_.size()),
                                           extr_vfromv(mus, 1, subcount));

    vector<FLOAT > matrixProfile(subcount, -1);
    for(auto value:nanmap)
    {
        matrixProfile[value-1] = NAN;
    }

    vector<int> matrixProfileIdx(subcount,NAN);

    FLOAT  cov_,corr_;

    for(int diag = minlag + 1;diag <= subcount;diag ++)
    {
        vector<FLOAT > temp_ts,temp_ts1;
        for(int i = diag;i <= diag + subseqlen -1;i++)
        {
            temp_ts.push_back(timeSeries_[i-1]-mu[diag-1]);
        }
        for(int i = 1;i <= subseqlen;i++)
        {
            temp_ts1.push_back(timeSeries_[i-1]-mu[0]);
        }

        cov_ = elementWiseMultiply_sum(temp_ts,temp_ts1);

        for(int row = 1; row <= subcount - diag + 1; row++)
        {
            int col = diag +row -1;
            if(row > 1)
            {
                cov_ = cov_ - dr_bwd[row-1]*dc_bwd[col-1] + dr_fwd[row-1]*dc_fwd[col-1];
            }

            corr_ = cov_ * invnorm[row-1]*invnorm[col-1];

            if(corr_ > matrixProfile[row-1])
            {
                matrixProfile[row-1] = corr_;
                matrixProfileIdx[row-1] = col;
            }
            if(corr_ > matrixProfile[col-1])
            {
                matrixProfile[col-1] = corr_;
                matrixProfileIdx[col-1] = row;
            }
        }
    }

    vector<FLOAT > discordsIdx = findDiscords(matrixProfile, minlag);

    vector<vector<int>> motifIdx(12, vector<int>(3,NAN));;
    findMotifs(timeSeries_,mu,invnorm,matrixProfile,matrixProfileIdx,subseqlen,minlag,motifIdx);

    vector<FLOAT > ttt;
    vector<FLOAT > matrixProfmaile;

    for(auto v:matrixProfile)
    {
        ttt.push_back(2*subseqlen*(1-v));
    }
    for(auto value: max_nv_Includenan(0,ttt))
    {
        matrixProfmaile.push_back(sqrt(value));
    }

    if(nanIDX.size() > 0.1)
    {
        for(auto value:nanIDX)
        {
            timeSeries_[value-1] = NAN;
        }
    }

    RETURN_MPX result;
    result.motifIdxs = motifIdx;
    result.matrixProfile = matrixProfmaile;
    result.discordIdx = discordsIdx;
    result.matrixProfileIdx = matrixProfileIdx;

    return result;

}

