#include "hip/hip_runtime.h"

#include "../alldef/matrix.cuh"
#include "iostream"
#include "vector"
#include "algorithm"
#include "cmath"
#include "chrono"
using namespace std;
bool LB_KK(const vector<FLOAT> &q, const FLOAT *U, const FLOAT *L, long long seqlen, FLOAT threshold_2,
           vector<FLOAT> &cb, FLOAT special_shared_vector[], FLOAT &lbk) {

    FLOAT  dist = 0.0;
    FLOAT temp_u;
    FLOAT temp_l;
    
    dist=special_shared_vector[0]+special_shared_vector[1];

    FLOAT temp_dist = 0;
    for (size_t i = 0; i < 3 ; i++) {

        FLOAT d = 0;
        temp_u = U[i];
        temp_l = L[i];
        if (q[i] > temp_u) {
            d = (q[i] - temp_u) * (q[i] - temp_u);
        } else if (q[i] < temp_l) {
            d = (q[i] - temp_l) * (q[i] - temp_l);
        }
        temp_dist += d;
    }

    for (size_t i = seqlen - 3; i < seqlen ; i++) {
        FLOAT d = 0;
        temp_u = U[i];
        temp_l = L[i];
        if (q[i] > temp_u) {
            d = (q[i] - temp_u) * (q[i] - temp_u);
        } else if (q[i] < temp_l) {
            d = (q[i] - temp_l) * (q[i] - temp_l);
        }
        temp_dist += d;
    }

    dist = MAX(temp_dist,dist);

    cb[0]=special_shared_vector[0];
    cb[1]=0;
    cb[2]=0;
    cb[seqlen-3]=special_shared_vector[1];
    cb[seqlen-2]=0;
    cb[seqlen-1]=0;

    for (size_t i = 3; i < seqlen - 3 ; i++) {
        FLOAT d = 0;
        temp_u = U[i];
        temp_l = L[i];
        if (q[i] > temp_u) {
            d = (q[i] - temp_u) * (q[i] - temp_u);
        } else if (q[i] < temp_l) {
            d = (q[i] - temp_l) * (q[i] - temp_l);
        }

        dist += d;
        if(threshold_2 < dist)
        {
            return 0;
        }
        cb[i] = d;
    }
    lbk = dist;
    return 1;
}

__device__ bool LB_KK(const FLOAT* q, const FLOAT *U, const FLOAT *L, long long seqlen, FLOAT threshold_2,
                      FLOAT *cb, FLOAT special_shared_vector[], FLOAT &lbk) {

    FLOAT  dist = 0.0;
    FLOAT temp_u;
    FLOAT temp_l;
    
    dist=special_shared_vector[0]+special_shared_vector[1];

    FLOAT temp_dist = 0;
    for (size_t i = 0; i < 3 ; i++) {

        FLOAT d = 0;
        temp_u = U[i];
        temp_l = L[i];
        if (q[i] > temp_u) {
            d = (q[i] - temp_u) * (q[i] - temp_u);
        } else if (q[i] < temp_l) {
            d = (q[i] - temp_l) * (q[i] - temp_l);
        }
        temp_dist += d;
    }

    for (size_t i = seqlen - 3; i < seqlen ; i++) {
        FLOAT d = 0;
        temp_u = U[i];
        temp_l = L[i];
        if (q[i] > temp_u) {
            d = (q[i] - temp_u) * (q[i] - temp_u);
        } else if (q[i] < temp_l) {
            d = (q[i] - temp_l) * (q[i] - temp_l);
        }
        temp_dist += d;
    }

    dist = MAX(temp_dist,dist);

    cb[0]=special_shared_vector[0];
    cb[1]=0;
    cb[2]=0;
    cb[seqlen-3]=special_shared_vector[1];
    cb[seqlen-2]=0;
    cb[seqlen-1]=0;

    for (size_t i = 3; i < seqlen - 3 ; i++) {
        FLOAT d = 0;
        temp_u = U[i];
        temp_l = L[i];
        if (q[i] > temp_u) {
            d = (q[i] - temp_u) * (q[i] - temp_u);
        } else if (q[i] < temp_l) {
            d = (q[i] - temp_l) * (q[i] - temp_l);
        }

        dist += d;
        if(threshold_2 < dist)
        {
            return 0;
        }
        cb[i] = d;
    }
    lbk = dist;
    return 1;
}

bool LB_KK_FIRST(const vector<FLOAT> &q, const vector<FLOAT> &t, const vector<FLOAT> &U, const vector<FLOAT> &L,
                 long long seqlen, FLOAT threshold_2, vector<FLOAT> &cb) {

    FLOAT  dist = 0.0;
    FLOAT temp_u;
    FLOAT temp_l;

    int m = seqlen;
    FLOAT d = 0;
    FLOAT x0 = t[0] ;
    FLOAT y0 = t[(m - 1 )] ;
    FLOAT dleft=DIST(x0, q[0]);
    FLOAT dright=DIST(y0, q[m - 1]);

    FLOAT x1 = (t[( 1)] );
    d = min(DIST(x1, q[0]), DIST(x0, q[1]));
    d = min(d, DIST(x1, q[1]));
    dleft+=d;

    FLOAT y1 = (t[(m - 2 )]);
    d = min(DIST(y1, q[m - 1]), DIST(y0, q[m - 2]) );
    d = min(d, DIST(y1, q[m - 2]));
    dright+=d;

    if (dleft+dright>=threshold_2){
        return false;
    }
    else{
        FLOAT x2 = (t[(2)]);
        d = min(DIST(x0, q[2]), DIST(x1, q[2]));
        d = min(d, DIST(x2, q[2]));
        d = min(d, DIST(x2, q[1]));
        d = min(d, DIST(x2, q[0]));
        dleft += d;

        FLOAT y2 = (t[(m - 3 )]);
        d = min(DIST(y0, q[m - 3]), DIST(y1, q[m - 3]));
        d = min(d, DIST(y2, q[m - 3]));
        d = min(d, DIST(y2, q[m - 2]));
        d = min(d, DIST(y2, q[m - 1]));
        dright += d;

        if (dleft+dright> threshold_2){
            return false;
        }
    }

    cb[0]=dleft;
    cb[1]=0;
    cb[2]=0;
    cb[seqlen-3]=dright;
    cb[seqlen-2]=0;
    cb[seqlen-1]=0;

    dist = dleft + dright;
    for (size_t i = 3; i < seqlen - 3; i++) {
        d = 0;
        temp_u = U[i];
        temp_l = L[i];
        if (q[i] > temp_u) {
            d = (q[i] - temp_u) * (q[i] - temp_u);
        } else if (q[i] < temp_l) {
            d = (q[i] - temp_l) * (q[i] - temp_l);
        }
        dist += d;
        if(threshold_2 < dist)
        {
            return false;
        }
        cb[i] = d;
    }
    return true;
}

bool LB_KK_FIRST(const FLOAT* q, const FLOAT* t, const FLOAT* U, const FLOAT* L,
                 long long seqlen, FLOAT threshold_2, vector<FLOAT> &cb) {

    FLOAT  dist = 0.0;
    FLOAT temp_u;
    FLOAT temp_l;

    int m = seqlen;
    FLOAT d = 0;
    FLOAT x0 = t[0] ;
    FLOAT y0 = t[(m - 1 )] ;
    FLOAT dleft=DIST(x0, q[0]);
    FLOAT dright=DIST(y0, q[m - 1]);

    FLOAT x1 = (t[( 1)] );
    d = min(DIST(x1, q[0]), DIST(x0, q[1]));
    d = min(d, DIST(x1, q[1]));
    dleft+=d;

    FLOAT y1 = (t[(m - 2 )]);
    d = min(DIST(y1, q[m - 1]), DIST(y0, q[m - 2]) );
    d = min(d, DIST(y1, q[m - 2]));
    dright+=d;

    if (dleft+dright>=threshold_2){
        return false;
    }
    else{
        FLOAT x2 = (t[(2)]);
        d = min(DIST(x0, q[2]), DIST(x1, q[2]));
        d = min(d, DIST(x2, q[2]));
        d = min(d, DIST(x2, q[1]));
        d = min(d, DIST(x2, q[0]));
        dleft += d;

        FLOAT y2 = (t[(m - 3 )]);
        d = min(DIST(y0, q[m - 3]), DIST(y1, q[m - 3]));
        d = min(d, DIST(y2, q[m - 3]));
        d = min(d, DIST(y2, q[m - 2]));
        d = min(d, DIST(y2, q[m - 1]));
        dright += d;

        if (dleft+dright> threshold_2){
            return false;
        }
    }

    cb[0]=dleft;
    cb[1]=0;
    cb[2]=0;
    cb[seqlen-3]=dright;
    cb[seqlen-2]=0;
    cb[seqlen-1]=0;

    dist = dleft + dright;
    for (size_t i = 3; i < seqlen - 3; i++) {
        d = 0;
        temp_u = U[i];
        temp_l = L[i];
        if (q[i] > temp_u) {
            d = (q[i] - temp_u) * (q[i] - temp_u);
        } else if (q[i] < temp_l) {
            d = (q[i] - temp_l) * (q[i] - temp_l);
        }
        dist += d;
        if(threshold_2 < dist)
        {
            return false;
        }
        cb[i] = d;
    }
    return true;
}