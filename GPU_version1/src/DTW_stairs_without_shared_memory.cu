#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include "GPU_parameters.h"
#include "matrix.cuh"
#define REGISTER_NUM 4
#define THREAD_NUM_PER_WARP 32

__device__ void
DTW_stairs_for_block_without_shared(FLOAT *Subject, FLOAT *cQuery, FLOAT &Dist, int m, FLOAT threshold, int w, FLOAT *q, FLOAT *t,int bl_size) {
    
    int num_per_bl = bl_size*bl_size;
    int tid = threadIdx.x;
    
    int num_tid = 32;
    
    q = cQuery;
    t = Subject;
    
    __syncthreads();
    int row_bias;
    int col_bias;
    int mid_tid;
    
    {

        mid_tid = 16;
        row_bias = (mid_tid-tid)*bl_size;
        col_bias = (tid - mid_tid)*bl_size;
    }

    int i_temp = row_bias;
    int j_temp = col_bias;

    FLOAT DTW_FIR[REGISTER_NUM*REGISTER_NUM];
    FLOAT DTW_SEC[REGISTER_NUM*REGISTER_NUM];
    for(int i = 0;i < num_per_bl;i++)
    {
        DTW_FIR[i] = INFINITY;
        DTW_SEC[i] = INFINITY;
        
    }
    if (tid == mid_tid) DTW_FIR[num_per_bl-1] = 0;

    FLOAT DTW_UP[REGISTER_NUM];
    FLOAT DTW_DOWN[REGISTER_NUM];
    
    for(int i = 0;i < bl_size;i++)
    {
        DTW_UP[i] = INFINITY;
        DTW_DOWN[i] = INFINITY;
    }

    FLOAT q0,t0,t1;

    int switch_for_stair = 0;
    bool mask;
    FLOAT d;

    int w_bias = ceil((w + 1.0)/bl_size);
    for(int step = 0; step < w_bias; step++){
        mask = switch_for_stair%2;
        switch_for_stair++;
        
        if(!mask)
        {
            for(int up_i = 0; up_i < bl_size; up_i++)
            {
                DTW_UP[up_i] = __shfl_up_sync(0xFFFFFFFF, DTW_SEC[num_per_bl - bl_size + up_i], 1, 32);
                
            }

            d = (i_temp >= 0 && j_temp >= 0) ?  DIST(q[i_temp],t[j_temp]) : INFINITY;
            DTW_FIR[0] = d + MIN(DTW_UP[0],MIN(DTW_SEC[bl_size - 1],DTW_FIR[num_per_bl-1]));

            i_temp++;

            for(int i = 1;i < bl_size;i++)
            {

                d = (i_temp >= 0 && j_temp >= 0) ?  DIST(q[i_temp],t[j_temp]) : INFINITY;
                DTW_FIR[i] = d + MIN(DTW_UP[i],MIN(DTW_UP[i-1],DTW_FIR[i-1]));

                i_temp++;
            }
            
            j_temp++;
            i_temp-=(bl_size);
            
            for(int i = 1;i < bl_size;i++)
            
            {

                d = (i_temp >= 0 && j_temp >= 0) ?  DIST(q[i_temp],t[j_temp]) : INFINITY;
                DTW_FIR[i*bl_size] =  d + MIN(DTW_SEC[(i+1)*bl_size - 1],MIN(DTW_SEC[(i)*bl_size - 1],DTW_FIR[(i-1)*bl_size]));

                i_temp++;
                
                for(int j = 1;j < bl_size;j++)
                
                {

                    d = (i_temp >= 0 && j_temp >= 0) ?  DIST(q[i_temp],t[j_temp]) : INFINITY;
                    DTW_FIR[i*bl_size + j] =  d + MIN(DTW_FIR[i*bl_size + j - 1],
                                                      MIN(DTW_FIR[(i-1)*bl_size + j - 1],DTW_FIR[(i-1)*bl_size + j]));

                    i_temp++;
                }
                j_temp++;
                i_temp-=(bl_size);
                
            }

        }
        else
        {
            for(int down_i = 0; down_i < bl_size; down_i++)
            {
                DTW_DOWN[down_i] = __shfl_down_sync(0xFFFFFFFF, DTW_FIR[(down_i+1)*bl_size - 1], 1, 32);
                
            }

            d = (i_temp >= 0 && j_temp >= 0) ?  DIST(q[i_temp],t[j_temp]) : INFINITY;
            DTW_SEC[0] = d + MIN(DTW_DOWN[0],MIN(DTW_FIR[num_per_bl - bl_size],
                                                                         DTW_SEC[num_per_bl - 1]));

            j_temp++;
            
            for(int i = 1;i < bl_size;i++)
            {

                d = (i_temp >= 0 && j_temp >= 0) ?  DIST(q[i_temp],t[j_temp]) : INFINITY;
                DTW_SEC[i*bl_size] = d + MIN(DTW_SEC[(i-1)*bl_size],MIN(DTW_DOWN[i-1],DTW_DOWN[i]));

                j_temp++;
            }
            
            i_temp++;
            j_temp-=(bl_size);
            
            for(int i = 1;i < bl_size;i++)
            {

                d = (i_temp >= 0 && j_temp >= 0) ?  DIST(q[i_temp],t[j_temp]) : INFINITY;
                DTW_SEC[i] =  d +
                        MIN(DTW_SEC[i - 1], MIN(DTW_FIR[num_per_bl - bl_size + i],DTW_FIR[num_per_bl - bl_size + i-1]));

                i_temp++;
            }
            i_temp-=(bl_size - 1);
            j_temp++;
            
            for(int i = 1;i < bl_size;i++)
                
            {
                for(int j = 1;j < bl_size;j++)
                    
                {

                    d = (i_temp >= 0 && j_temp >= 0) ?  DIST(q[i_temp],t[j_temp]) : INFINITY;
                    DTW_SEC[i*bl_size + j] =  d + MIN(DTW_SEC[i*bl_size + j - 1],
                                                                              MIN(DTW_SEC[(i-1)*bl_size + j - 1],DTW_SEC[(i-1)*bl_size + j]));

                    i_temp++;
                }
                j_temp++;
                i_temp-=(bl_size - 1);
                
            }

            j_temp-=(bl_size);
            i_temp +=(bl_size - 1);

        }

    }

    __syncthreads();

    FLOAT x,y,z;
    
    for(int step = w_bias; step < ceil(2.0*m/bl_size) - w_bias - 1; step++){
        mask = switch_for_stair%2;
        switch_for_stair++;
        
        if(!mask)
        {
            for(int up_i = 0; up_i < bl_size; up_i++)
            {
                DTW_UP[up_i] = __shfl_up_sync(0xFFFFFFFF, DTW_SEC[num_per_bl - bl_size + up_i], 1, 32);
                
            }

            d = (i_temp - j_temp > w ||  j_temp - i_temp> w) ? INFINITY : DIST(q[i_temp],t[j_temp]);

            DTW_FIR[0] = d + MIN(DTW_UP[0],MIN(DTW_SEC[bl_size - 1],DTW_FIR[num_per_bl-1]));

            i_temp++;
            
            for(int i = 1;i < bl_size;i++)
            {

                d = (i_temp - j_temp > w ||  j_temp - i_temp> w) ? INFINITY : DIST(q[i_temp],t[j_temp]);
                DTW_FIR[i] = d + MIN(DTW_UP[i],MIN(DTW_UP[i-1],DTW_FIR[i-1]));

                i_temp++;
            }
            
            j_temp++;
            i_temp-=(bl_size);
            
            for(int i = 1;i < bl_size;i++)
                
            {

               d = (i_temp - j_temp > w ||  j_temp - i_temp> w) ? INFINITY : DIST(q[i_temp],t[j_temp]);

                DTW_FIR[i*bl_size] =  d +
                        MIN(DTW_SEC[(i+1)*bl_size - 1],MIN(DTW_SEC[(i)*bl_size - 1],DTW_FIR[(i-1)*bl_size]));

                i_temp++;
                
                for(int j = 1;j < bl_size;j++)
                    
                {

                    d = (i_temp - j_temp > w ||  j_temp - i_temp> w) ? INFINITY : DIST(q[i_temp],t[j_temp]);

                    DTW_FIR[i*bl_size + j] =  d +
                            MIN(DTW_FIR[i*bl_size + j - 1],MIN(DTW_FIR[(i-1)*bl_size + j - 1],DTW_FIR[(i-1)*bl_size + j]));

                    i_temp++;
                }
                j_temp++;
                i_temp-=(bl_size);
                
            }
        }
            
        else
        {
            for(int down_i = 0; down_i < bl_size; down_i++)
            {
                DTW_DOWN[down_i] = __shfl_down_sync(0xFFFFFFFF, DTW_FIR[(down_i+1)*bl_size - 1], 1, 32);
                
            }

            d = (i_temp - j_temp > w ||  j_temp - i_temp> w) ? INFINITY  :DIST(q[i_temp],t[j_temp]);

            DTW_SEC[0] = d + MIN(DTW_DOWN[0],MIN(DTW_FIR[num_per_bl - bl_size],
                                                                         DTW_SEC[num_per_bl - 1]));

            j_temp++;

            for(int i = 1;i < bl_size;i++)
            {
                d = (i_temp - j_temp > w ||  j_temp - i_temp> w) ? INFINITY : DIST(q[i_temp],t[j_temp]);

                DTW_SEC[i*bl_size] = d + MIN(DTW_SEC[(i-1)*bl_size],MIN(DTW_DOWN[i-1],DTW_DOWN[i]));

                j_temp++;
            }
            
            i_temp++;
            j_temp-=(bl_size);
            
            for(int i = 1;i < bl_size;i++)
            {
                d = (i_temp - j_temp > w ||  j_temp - i_temp> w) ? INFINITY : DIST(q[i_temp],t[j_temp]);

                DTW_SEC[i] =  d +
                              MIN(DTW_SEC[i - 1], MIN(DTW_FIR[num_per_bl - bl_size + i],DTW_FIR[num_per_bl - bl_size + i-1]));

                i_temp++;
            }
            i_temp-=(bl_size - 1);
            j_temp++;
            
            for(int i = 1;i < bl_size;i++)
                
            {
                for(int j = 1;j < bl_size;j++)
                    
                {
                    d = (i_temp - j_temp > w ||  j_temp - i_temp> w) ? INFINITY : DIST(q[i_temp],t[j_temp]);

                    DTW_SEC[i*bl_size + j] = d + MIN(DTW_SEC[i*bl_size + j - 1],
                                                                              MIN(DTW_SEC[(i-1)*bl_size + j - 1],DTW_SEC[(i-1)*bl_size + j]));

                    i_temp++;
                }
                j_temp++;
                i_temp-=(bl_size - 1);
                
            }
            j_temp-=bl_size;
            i_temp +=(bl_size - 1);
        }

    }

    __syncthreads();

    for(int step = ceil(2.0*m/bl_size) - w_bias - 1; step <  ceil(2.0*m/bl_size); step++){
        mask = switch_for_stair%2;
        switch_for_stair++;
        
        if(!mask)
        {
            for(int up_i = 0; up_i < bl_size; up_i++)
            {
                DTW_UP[up_i] = __shfl_up_sync(0xFFFFFFFF, DTW_SEC[num_per_bl - bl_size + up_i], 1, 32);
                
            }

            d = (i_temp < m && j_temp < m) ?  DIST(q[i_temp],t[j_temp]) : INFINITY;

            DTW_FIR[0] = d + MIN(DTW_UP[0],MIN(DTW_SEC[bl_size - 1],DTW_FIR[num_per_bl-1]));

            if(tid == mid_tid && i_temp == m - 1 && j_temp == m - 1)
            {
                Dist = sqrt(DTW_FIR[0]);

            }
            i_temp++;
            
            for(int i = 1;i < bl_size;i++)
            {
                d = (i_temp < m && j_temp < m) ?  DIST(q[i_temp],t[j_temp]) : INFINITY;

                DTW_FIR[i] = d + MIN(DTW_UP[i],MIN(DTW_UP[i-1],DTW_FIR[i-1]));

                if(tid == mid_tid && i_temp == m - 1 && j_temp == m - 1)
                {
                    Dist = sqrt(DTW_FIR[i]);

                }
                i_temp++;
            }
            
            j_temp++;
            i_temp-=(bl_size);
            
            for(int i = 1;i < bl_size;i++)
                
            {
                d = (i_temp < m && j_temp < m) ?  DIST(q[i_temp],t[j_temp]) : INFINITY;

                DTW_FIR[i*bl_size] =  d + MIN(DTW_SEC[(i+1)*bl_size - 1],MIN(DTW_SEC[(i)*bl_size - 1],DTW_FIR[(i-1)*bl_size]));

                if(tid == mid_tid && i_temp == m - 1  && j_temp == m - 1 )
                {
                    Dist = sqrt(DTW_FIR[i*bl_size]);

                }
                i_temp++;
                
                for(int j = 1;j < bl_size;j++)
                    
                {
                    d = (i_temp < m && j_temp < m) ?  DIST(q[i_temp],t[j_temp]) : INFINITY;

                    DTW_FIR[i*bl_size + j] =  d + MIN(DTW_FIR[i*bl_size + j - 1],
                                                                              MIN(DTW_FIR[(i-1)*bl_size + j - 1],DTW_FIR[(i-1)*bl_size + j]));

                    if(tid == mid_tid && i_temp == m - 1 && j_temp == m - 1)
                    {
                        Dist = sqrt(DTW_FIR[i*bl_size + j]);

                    }
                    i_temp++;
                }
                j_temp++;
                i_temp-=(bl_size);
                
            }

        }
            
        else
        {
            for(int down_i = 0; down_i < bl_size; down_i++)
            {
                DTW_DOWN[down_i] = __shfl_down_sync(0xFFFFFFFF, DTW_FIR[(down_i+1)*bl_size - 1], 1, 32);
                
            }
            d = (i_temp < m && j_temp < m) ?  DIST(q[i_temp],t[j_temp]) : INFINITY;

            DTW_SEC[0] = d + MIN(DTW_DOWN[0],MIN(DTW_FIR[num_per_bl - bl_size],
                                                                         DTW_SEC[num_per_bl - 1]));

            j_temp++;
            
            for(int i = 1;i < bl_size;i++)
            {
                d = (i_temp < m && j_temp < m) ?  DIST(q[i_temp],t[j_temp]) : INFINITY;

                DTW_SEC[i*bl_size] = d + MIN(DTW_SEC[(i-1)*bl_size],
                                                                     MIN(DTW_DOWN[i-1],DTW_DOWN[i]));

                j_temp++;
            }
            
            i_temp++;
            j_temp-=(bl_size);
            
            for(int i = 1;i < bl_size;i++)
            {
                d = (i_temp < m && j_temp < m) ?  DIST(q[i_temp],t[j_temp]) : INFINITY;

                DTW_SEC[i] =  d +
                              MIN(DTW_SEC[i - 1], MIN(DTW_FIR[num_per_bl - bl_size + i],DTW_FIR[num_per_bl - bl_size + i-1]));
                if(i_temp <0 || j_temp < 0)
                {
                    printf("fuck in 482 i_temp = %d,j_temp = %d,tid = %d,step = %d\n",i_temp,j_temp,tid,step);
                }

                i_temp++;
            }
            j_temp++;
            i_temp-=(bl_size - 1);
            
            for(int i = 1;i < bl_size;i++)
                
            {
                for(int j = 1;j < bl_size;j++)
                    
                {
                    d = (i_temp < m && j_temp < m) ?  DIST(q[i_temp],t[j_temp]) : INFINITY;

                    DTW_SEC[i*bl_size + j] =  d + MIN(DTW_SEC[i*bl_size + j - 1],
                                                                              MIN(DTW_SEC[(i-1)*bl_size + j - 1],DTW_SEC[(i-1)*bl_size + j]));
                    if( i_temp <0 || j_temp < 0)
                    {
                        printf("fuck in 499 i_temp = %d,j_temp = %d,tid = %d,step = %d\n",i_temp,j_temp,tid,step);
                    }

                    i_temp++;
                }
                j_temp++;
                i_temp-=(bl_size - 1);
                
            }
            j_temp-=bl_size;
            i_temp +=(bl_size - 1);

        }
    }
}

__device__ void
DTW_stairs_without_shared(FLOAT *Subject, FLOAT *cQuery, FLOAT &Dist, int m, FLOAT threshold_2, int w, FLOAT *q, FLOAT *t) {

    int tid = threadIdx.x%THREAD_NUM_PER_WARP;
    
    int num_tid = THREAD_NUM_PER_WARP;
    
    q =cQuery;
    t = Subject;

    FLOAT DTW_FIR;
    
    if (tid == 16) {
        DTW_FIR = 0;  
    } else {
        DTW_FIR = INFINITY;
    }
    FLOAT DTW_SEC = INFINITY;

    FLOAT DTW_UP = 0;
    FLOAT DTW_DOWN = 0;
    
    int row_bias = 16 - tid;
    int col_bias = tid - 16;

    int i_temp = row_bias;
    int j_temp = col_bias;
    size_t vote;
    
    bool flag_pruning = 0;

    FLOAT d;
    int switch_for_stair = 0;
    bool mask;
    for(int i = 0;i < w;i++){
        mask = switch_for_stair%2;
        switch_for_stair++;
        
        if(!mask)
        {
            DTW_UP = __shfl_up_sync(0xFFFFFFFF, DTW_SEC, 1, 32);
            d = ((i_temp >= 0) && (j_temp >= 0)) ? DIST(q[i_temp],t[j_temp]) : INFINITY;
            DTW_FIR = d + MIN(DTW_FIR,MIN(DTW_SEC,DTW_UP));

            j_temp++;
        }
        
        else
        {
            DTW_DOWN = __shfl_down_sync(0xFFFFFFFF, DTW_FIR, 1, 32);
            
            d = ((i_temp >= 0) && (j_temp >= 0)) ? DIST(q[i_temp],t[j_temp]) : INFINITY;
            DTW_SEC = d + MIN(DTW_FIR,MIN(DTW_SEC,DTW_DOWN));

            i_temp++;
        }
    }

    int cb_index ;
    FLOAT cb_temp;
    FLOAT cb_temp1;
    FLOAT cb_temp2;
    for(int i = w;i < 2*m - 1 - w;i++){

        mask = switch_for_stair%2;
        switch_for_stair++;
        
        if(!mask)
        {
            DTW_UP = __shfl_up_sync(0xFFFFFFFF, DTW_SEC, 1, 32);
            
            d = (i_temp - j_temp > w ||  j_temp - i_temp> w) ? INFINITY  :DIST(q[i_temp],t[j_temp]);

            if(i_temp - j_temp ==  w)
            {
                DTW_FIR = d + MIN(DTW_FIR,DTW_SEC);
            }
            else
            {
                DTW_FIR = d + MIN(DTW_FIR,MIN(DTW_SEC,DTW_UP));
            }

            j_temp++;

        }
            
        else
        {
            DTW_DOWN = __shfl_down_sync(0xFFFFFFFF, DTW_FIR, 1, 32);
            
            d = (i_temp - j_temp > w ||  j_temp - i_temp> w) ? INFINITY  :DIST(q[i_temp],t[j_temp]);

            if(j_temp - i_temp ==  w)
            {
                DTW_SEC = d + MIN(DTW_FIR,DTW_SEC);
            }
            else
            {
                DTW_SEC = d + MIN(DTW_FIR,MIN(DTW_SEC,DTW_DOWN));
            }

            i_temp++;

        }

    }

    if(!flag_pruning)
    {
        for(int i = 2*m  - 1- w;i < 2*m - 1;i++){
            mask = switch_for_stair%2;
            switch_for_stair++;
            
            if(!mask)
            {
                DTW_UP = __shfl_up_sync(0xFFFFFFFF, DTW_SEC, 1, 32);
                
                d = (i_temp < m && j_temp < m) ?  DIST(q[i_temp],t[j_temp]) : INFINITY;
                DTW_FIR = d + MIN(DTW_FIR,MIN(DTW_SEC,DTW_UP));

                j_temp++;
            }
                
            else
            {
                DTW_DOWN = __shfl_down_sync(0xFFFFFFFF, DTW_FIR, 1, 32);
                
                d = (i_temp < m && j_temp < m) ?  DIST(q[i_temp],t[j_temp]) : INFINITY;
                DTW_SEC = d + MIN(DTW_FIR,MIN(DTW_SEC,DTW_DOWN));

                i_temp++;
            }

        }

        if(tid == 16)
        {
            Dist = sqrt(DTW_FIR);

        }
    }

}

