#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include "GPU_parameters.h"
#include "matrix.cuh"
#define REGISTER_NUM 2
#define THREAD_NUM_PER_WARP 32

__device__ void
DTW_stairs_for_block_without_shared(FLOAT *Subject, FLOAT *cQuery, FLOAT &Dist, int m, FLOAT threshold, int w,
                                    int bl_size, FLOAT cb[], FLOAT threshold_2) {
    
    bl_size = REGISTER_NUM;

    int num_per_bl = bl_size*bl_size;
    int tid = threadIdx.x;
    
    int num_tid = 32;
    size_t vote;
    
    FLOAT *q =cQuery;
    FLOAT *t = Subject;
    
    __syncthreads();
    int row_bias;
    int col_bias;
    int mid_tid;
    
    {

        mid_tid = 16;
        row_bias = (mid_tid-tid)*bl_size;
        col_bias = (tid - mid_tid)*bl_size;
    }

    int i_temp = row_bias;
    int j_temp = col_bias;

    FLOAT DTW_FIR[REGISTER_NUM*REGISTER_NUM];
    FLOAT DTW_SEC[REGISTER_NUM*REGISTER_NUM];
    for(int i = 0;i < num_per_bl;i++)
    {
        DTW_FIR[i] = INFINITY;
        DTW_SEC[i] = INFINITY;
        
    }
    if (tid == mid_tid) DTW_FIR[num_per_bl-1] = 0;

    FLOAT DTW_UP[REGISTER_NUM];
    FLOAT DTW_DOWN[REGISTER_NUM];
    
    for(int i = 0;i < bl_size;i++)
    {
        DTW_UP[i] = INFINITY;
        DTW_DOWN[i] = INFINITY;
    }

    FLOAT q0,t0,t1;
    int cb_index ;
    FLOAT cb_temp;
    int switch_for_stair = 0;
    bool mask;
    FLOAT d;
    bool flag_pruning = false;

    int target_j_temp = (m/2) - (m/2)%bl_size - 1;

    int w_bias = ceil((w + 1.0)/bl_size);
    for(int step = 0; step < w_bias; step++){
        mask = switch_for_stair%2;
        switch_for_stair++;
        
        if(!mask)
        {
            for(int up_i = 0; up_i < bl_size; up_i++)
            {
                DTW_UP[up_i] = __shfl_up_sync(0xFFFFFFFF, DTW_SEC[num_per_bl - bl_size + up_i], 1, 32);
                
            }

            d = (i_temp >= 0 && j_temp >= 0) ?  DIST(q[i_temp],t[j_temp]) : INFINITY;
            DTW_FIR[0] = d + MIN(DTW_UP[0],MIN(DTW_SEC[bl_size - 1],DTW_FIR[num_per_bl-1]));

            i_temp++;

            for(int i = 1;i < bl_size;i++)
            {

                d = (i_temp >= 0 && j_temp >= 0) ?  DIST(q[i_temp],t[j_temp]) : INFINITY;
                DTW_FIR[i] = d + MIN(DTW_UP[i],MIN(DTW_UP[i-1],DTW_FIR[i-1]));

                i_temp++;
            }
            
            j_temp++;
            i_temp-=(bl_size);
            
            for(int i = 1;i < bl_size;i++)
            
            {

                d = (i_temp >= 0 && j_temp >= 0) ?  DIST(q[i_temp],t[j_temp]) : INFINITY;
                DTW_FIR[i*bl_size] =  d + MIN(DTW_SEC[(i+1)*bl_size - 1],MIN(DTW_SEC[(i)*bl_size - 1],DTW_FIR[(i-1)*bl_size]));

                i_temp++;
                
                for(int j = 1;j < bl_size;j++)
                
                {

                    d = (i_temp >= 0 && j_temp >= 0) ?  DIST(q[i_temp],t[j_temp]) : INFINITY;
                    DTW_FIR[i*bl_size + j] =  d + MIN(DTW_FIR[i*bl_size + j - 1],
                                                      MIN(DTW_FIR[(i-1)*bl_size + j - 1],DTW_FIR[(i-1)*bl_size + j]));

                    i_temp++;
                }
                j_temp++;
                i_temp-=(bl_size);
                
            }

        }
        else
        {
            for(int down_i = 0; down_i < bl_size; down_i++)
            {
                DTW_DOWN[down_i] = __shfl_down_sync(0xFFFFFFFF, DTW_FIR[(down_i+1)*bl_size - 1], 1, 32);
                
            }

            d = (i_temp >= 0 && j_temp >= 0) ?  DIST(q[i_temp],t[j_temp]) : INFINITY;
            DTW_SEC[0] = d + MIN(DTW_DOWN[0],MIN(DTW_FIR[num_per_bl - bl_size],
                                                                         DTW_SEC[num_per_bl - 1]));

            j_temp++;
            
            for(int i = 1;i < bl_size;i++)
            {

                d = (i_temp >= 0 && j_temp >= 0) ?  DIST(q[i_temp],t[j_temp]) : INFINITY;
                DTW_SEC[i*bl_size] = d + MIN(DTW_SEC[(i-1)*bl_size],MIN(DTW_DOWN[i-1],DTW_DOWN[i]));

                j_temp++;
            }
            
            i_temp++;
            j_temp-=(bl_size);
            
            for(int i = 1;i < bl_size;i++)
            {

                d = (i_temp >= 0 && j_temp >= 0) ?  DIST(q[i_temp],t[j_temp]) : INFINITY;
                DTW_SEC[i] =  d +
                        MIN(DTW_SEC[i - 1], MIN(DTW_FIR[num_per_bl - bl_size + i],DTW_FIR[num_per_bl - bl_size + i-1]));

                i_temp++;
            }
            i_temp-=(bl_size - 1);
            j_temp++;
            
            for(int i = 1;i < bl_size;i++)
                
            {
                for(int j = 1;j < bl_size;j++)
                    
                {

                    d = (i_temp >= 0 && j_temp >= 0) ?  DIST(q[i_temp],t[j_temp]) : INFINITY;
                    DTW_SEC[i*bl_size + j] =  d + MIN(DTW_SEC[i*bl_size + j - 1],
                                                                              MIN(DTW_SEC[(i-1)*bl_size + j - 1],DTW_SEC[(i-1)*bl_size + j]));

                    i_temp++;
                }
                j_temp++;
                i_temp-=(bl_size - 1);
                
            }

            j_temp-=(bl_size);
            i_temp +=(bl_size - 1);

        }

    }

    __syncthreads();

    FLOAT x,y,z;
    
    for(int step = w_bias; step < ceil(2.0*m/bl_size) - w_bias - 1; step++){
        mask = switch_for_stair%2;
        switch_for_stair++;
        
        if(!mask)
        {
            for(int up_i = 0; up_i < bl_size; up_i++)
            {
                DTW_UP[up_i] = __shfl_up_sync(0xFFFFFFFF, DTW_SEC[num_per_bl - bl_size + up_i], 1, 32);
                
            }

            d = (i_temp - j_temp > w ||  j_temp - i_temp> w) ? INFINITY : DIST(q[i_temp],t[j_temp]);

            DTW_FIR[0] = d + MIN(DTW_UP[0],MIN(DTW_SEC[bl_size - 1],DTW_FIR[num_per_bl-1]));

            i_temp++;
            
            for(int i = 1;i < bl_size;i++)
            {
                d = (i_temp - j_temp > w ||  j_temp - i_temp> w) ? INFINITY : DIST(q[i_temp],t[j_temp]);
                DTW_FIR[i] = d + MIN(DTW_UP[i],MIN(DTW_UP[i-1],DTW_FIR[i-1]));

                i_temp++;
            }

            j_temp++;
            i_temp-=(bl_size);
            
            for(int i = 1;i < bl_size;i++)
                
            {

               d = (i_temp - j_temp > w ||  j_temp - i_temp> w) ? INFINITY : DIST(q[i_temp],t[j_temp]);

                DTW_FIR[i*bl_size] =  d +
                        MIN(DTW_SEC[(i+1)*bl_size - 1],MIN(DTW_SEC[(i)*bl_size - 1],DTW_FIR[(i-1)*bl_size]));

                i_temp++;
                
                for(int j = 1;j < bl_size;j++)
                    
                {

                    d = (i_temp - j_temp > w ||  j_temp - i_temp> w) ? INFINITY : DIST(q[i_temp],t[j_temp]);

                    DTW_FIR[i*bl_size + j] =  d +
                            MIN(DTW_FIR[i*bl_size + j - 1],MIN(DTW_FIR[(i-1)*bl_size + j - 1],DTW_FIR[(i-1)*bl_size + j]));

                    bool ifcb = (tid == 16) && (j_temp == target_j_temp) && (j == bl_size - 1) && (i == bl_size - 1) ;
                    ifcb = __shfl_sync(0x1F, ifcb, 16);
                    if(ifcb)
                    {
                        cb_index =(w + j_temp)/CB_LEN+1;
                        cb_temp =cb[cb_index];
                        bool all_greater = true;
                        for (int index_of_fir = 0; index_of_fir < bl_size; ++index_of_fir) {
                            if (DTW_FIR[index_of_fir] <= threshold_2 - cb_temp) {
                                all_greater = false;
                                break;
                            }
                        }

                        for (int index_of_fir = 1; index_of_fir < bl_size; ++index_of_fir) {
                            int left_index = index_of_fir*bl_size;
                            if (DTW_FIR[left_index] <= threshold_2 - cb_temp) {
                                all_greater = false;
                                break;
                            }
                        }

                        vote = __ballot_sync(0xFFFFFFFF, all_greater);
                        if (vote == 0xFFFFFFFF) {
                            if(tid == 16)
                            {
                                Dist = INFINITY;

                            }
                            flag_pruning = true;
                            return;
                        }
                    }
                    i_temp++;
                }
                j_temp++;
                i_temp-=(bl_size);
                
            }
        }
            
        else
        {
            for(int down_i = 0; down_i < bl_size; down_i++)
            {
                DTW_DOWN[down_i] = __shfl_down_sync(0xFFFFFFFF, DTW_FIR[(down_i+1)*bl_size - 1], 1, 32);
                
            }

            d = (i_temp - j_temp > w ||  j_temp - i_temp> w) ? INFINITY  :DIST(q[i_temp],t[j_temp]);

            DTW_SEC[0] = d + MIN(DTW_DOWN[0],MIN(DTW_FIR[num_per_bl - bl_size],
                                                                         DTW_SEC[num_per_bl - 1]));

            j_temp++;

            for(int i = 1;i < bl_size;i++)
            {
                d = (i_temp - j_temp > w ||  j_temp - i_temp> w) ? INFINITY : DIST(q[i_temp],t[j_temp]);

                DTW_SEC[i*bl_size] = d + MIN(DTW_SEC[(i-1)*bl_size],MIN(DTW_DOWN[i-1],DTW_DOWN[i]));

                j_temp++;
            }
            
            i_temp++;
            j_temp-=(bl_size);
            
            for(int i = 1;i < bl_size;i++)
            {
                d = (i_temp - j_temp > w ||  j_temp - i_temp> w) ? INFINITY : DIST(q[i_temp],t[j_temp]);

                DTW_SEC[i] =  d +
                              MIN(DTW_SEC[i - 1], MIN(DTW_FIR[num_per_bl - bl_size + i],DTW_FIR[num_per_bl - bl_size + i-1]));

                i_temp++;
            }
            i_temp-=(bl_size - 1);
            j_temp++;
            
            for(int i = 1;i < bl_size;i++)
                
            {
                for(int j = 1;j < bl_size;j++)
                    
                {
                    d = (i_temp - j_temp > w ||  j_temp - i_temp> w) ? INFINITY : DIST(q[i_temp],t[j_temp]);

                    DTW_SEC[i*bl_size + j] = d + MIN(DTW_SEC[i*bl_size + j - 1],
                                                                              MIN(DTW_SEC[(i-1)*bl_size + j - 1],DTW_SEC[(i-1)*bl_size + j]));

                    i_temp++;
                }
                j_temp++;
                i_temp-=(bl_size - 1);
                
            }
            j_temp-=bl_size;
            i_temp +=(bl_size - 1);
        }

    }

    __syncthreads();

    if(!flag_pruning)
    {
    for(int step = ceil(2.0*m/bl_size) - w_bias - 1; step <  ceil(2.0*m/bl_size); step++){
        mask = switch_for_stair%2;
        switch_for_stair++;
        
        if(!mask)
        {
            for(int up_i = 0; up_i < bl_size; up_i++)
            {
                DTW_UP[up_i] = __shfl_up_sync(0xFFFFFFFF, DTW_SEC[num_per_bl - bl_size + up_i], 1, 32);
                
            }

            d = (i_temp < m && j_temp < m) ?  DIST(q[i_temp],t[j_temp]) : INFINITY;

            DTW_FIR[0] = d + MIN(DTW_UP[0],MIN(DTW_SEC[bl_size - 1],DTW_FIR[num_per_bl-1]));

            if(tid == mid_tid && i_temp == m - 1 && j_temp == m - 1)
            {
                Dist = sqrt(DTW_FIR[0]);

            }
            i_temp++;
            
            for(int i = 1;i < bl_size;i++)
            {
                d = (i_temp < m && j_temp < m) ?  DIST(q[i_temp],t[j_temp]) : INFINITY;

                DTW_FIR[i] = d + MIN(DTW_UP[i],MIN(DTW_UP[i-1],DTW_FIR[i-1]));

                if(tid == mid_tid && i_temp == m - 1 && j_temp == m - 1)
                {
                    Dist = sqrt(DTW_FIR[i]);

                }
                i_temp++;
            }
            
            j_temp++;
            i_temp-=(bl_size);
            
            for(int i = 1;i < bl_size;i++)
                
            {
                d = (i_temp < m && j_temp < m) ?  DIST(q[i_temp],t[j_temp]) : INFINITY;

                DTW_FIR[i*bl_size] =  d + MIN(DTW_SEC[(i+1)*bl_size - 1],MIN(DTW_SEC[(i)*bl_size - 1],DTW_FIR[(i-1)*bl_size]));

                if(tid == mid_tid && i_temp == m - 1  && j_temp == m - 1 )
                {
                    Dist = sqrt(DTW_FIR[i*bl_size]);

                }
                i_temp++;
                
                for(int j = 1;j < bl_size;j++)
                    
                {
                    d = (i_temp < m && j_temp < m) ?  DIST(q[i_temp],t[j_temp]) : INFINITY;

                    DTW_FIR[i*bl_size + j] =  d + MIN(DTW_FIR[i*bl_size + j - 1],
                                                                              MIN(DTW_FIR[(i-1)*bl_size + j - 1],DTW_FIR[(i-1)*bl_size + j]));

                    if(tid == mid_tid && i_temp == m - 1 && j_temp == m - 1)
                    {
                        Dist = sqrt(DTW_FIR[i*bl_size + j]);

                    }
                    i_temp++;
                }
                j_temp++;
                i_temp-=(bl_size);
                
            }

        }
            
        else
        {
            for(int down_i = 0; down_i < bl_size; down_i++)
            {
                DTW_DOWN[down_i] = __shfl_down_sync(0xFFFFFFFF, DTW_FIR[(down_i+1)*bl_size - 1], 1, 32);
                
            }
            d = (i_temp < m && j_temp < m) ?  DIST(q[i_temp],t[j_temp]) : INFINITY;

            DTW_SEC[0] = d + MIN(DTW_DOWN[0],MIN(DTW_FIR[num_per_bl - bl_size],
                                                                         DTW_SEC[num_per_bl - 1]));

            j_temp++;
            
            for(int i = 1;i < bl_size;i++)
            {
                d = (i_temp < m && j_temp < m) ?  DIST(q[i_temp],t[j_temp]) : INFINITY;

                DTW_SEC[i*bl_size] = d + MIN(DTW_SEC[(i-1)*bl_size],
                                                                     MIN(DTW_DOWN[i-1],DTW_DOWN[i]));

                j_temp++;
            }
            
            i_temp++;
            j_temp-=(bl_size);
            
            for(int i = 1;i < bl_size;i++)
            {
                d = (i_temp < m && j_temp < m) ?  DIST(q[i_temp],t[j_temp]) : INFINITY;

                DTW_SEC[i] =  d +
                              MIN(DTW_SEC[i - 1], MIN(DTW_FIR[num_per_bl - bl_size + i],DTW_FIR[num_per_bl - bl_size + i-1]));

                i_temp++;
            }
            j_temp++;
            i_temp-=(bl_size - 1);
            
            for(int i = 1;i < bl_size;i++)
                
            {
                for(int j = 1;j < bl_size;j++)
                    
                {
                    d = (i_temp < m && j_temp < m) ?  DIST(q[i_temp],t[j_temp]) : INFINITY;

                    DTW_SEC[i*bl_size + j] =  d + MIN(DTW_SEC[i*bl_size + j - 1],
                                                                              MIN(DTW_SEC[(i-1)*bl_size + j - 1],DTW_SEC[(i-1)*bl_size + j]));

                    i_temp++;
                }
                j_temp++;
                i_temp-=(bl_size - 1);
                
            }
            j_temp-=bl_size;
            i_temp +=(bl_size - 1);

        }
    }
    }
}

__device__ void
DTW_stairs_without_shared(FLOAT *Subject, FLOAT *cQuery, FLOAT &Dist, int m, FLOAT threshold_2, int w, const FLOAT cb[]) {

    int tid = threadIdx.x%THREAD_NUM_PER_WARP;
    
    int num_tid = THREAD_NUM_PER_WARP;
    
    FLOAT *q =cQuery;
    FLOAT *t = Subject;

    FLOAT DTW_FIR;
    
    if (tid == 16) {
        DTW_FIR = 0;  
    } else {
        DTW_FIR = INFINITY;
    }
    FLOAT DTW_SEC = INFINITY;

    FLOAT DTW_UP = 0;
    FLOAT DTW_DOWN = 0;
    
    int row_bias = 16 - tid;
    int col_bias = tid - 16;

    int i_temp = row_bias;
    int j_temp = col_bias;
    size_t vote;
    
    bool flag_pruning = 0;

    FLOAT d;
    int switch_for_stair = 0;
    bool mask;
    for(int i = 0;i < w;i++){
        mask = switch_for_stair%2;
        switch_for_stair++;
        
        if(!mask)
        {
            DTW_UP = __shfl_up_sync(0xFFFFFFFF, DTW_SEC, 1, 32);
            d = ((i_temp >= 0) && (j_temp >= 0)) ? DIST(q[i_temp],t[j_temp]) : INFINITY;
            DTW_FIR = d + MIN(DTW_FIR,MIN(DTW_SEC,DTW_UP));

            j_temp++;
        }
        
        else
        {
            DTW_DOWN = __shfl_down_sync(0xFFFFFFFF, DTW_FIR, 1, 32);
            
            d = ((i_temp >= 0) && (j_temp >= 0)) ? DIST(q[i_temp],t[j_temp]) : INFINITY;
            DTW_SEC = d + MIN(DTW_FIR,MIN(DTW_SEC,DTW_DOWN));

            i_temp++;
        }
    }

    int cb_index ;
    FLOAT cb_temp;
    for(int i = w;i < 2*m - 1 - w;i++){

        mask = switch_for_stair%2;
        switch_for_stair++;
        
        if(!mask)
        {
            DTW_UP = __shfl_up_sync(0xFFFFFFFF, DTW_SEC, 1, 32);
            
            d = (i_temp - j_temp > w ||  j_temp - i_temp> w) ? INFINITY  :DIST(q[i_temp],t[j_temp]);

            if(i_temp - j_temp ==  w)
            {
                DTW_FIR = d + MIN(DTW_FIR,DTW_SEC);
            }
            else
            {
                DTW_FIR = d + MIN(DTW_FIR,MIN(DTW_SEC,DTW_UP));
            }

            bool ifcb = (tid == 16) && (j_temp == m/2) ;
            ifcb = __shfl_sync(0x1F, ifcb, 16);
            if(ifcb)
            {
                cb_index =(w + j_temp)/CB_LEN+1;
                cb_temp =cb[cb_index];
                vote = __ballot_sync(0xFFFFFFFF, DTW_FIR > threshold_2 - cb_temp);
                if (vote == 0xFFFFFFFF) {
                    if(tid == 16)
                    {
                        Dist = INFINITY;

                    }
                    return;
                    flag_pruning = true;
                }
            }

            j_temp++;

        }
            
        else
        {
            DTW_DOWN = __shfl_down_sync(0xFFFFFFFF, DTW_FIR, 1, 32);
            
            d = (i_temp - j_temp > w ||  j_temp - i_temp> w) ? INFINITY  :DIST(q[i_temp],t[j_temp]);

            if(j_temp - i_temp ==  w)
            {
                DTW_SEC = d + MIN(DTW_FIR,DTW_SEC);
            }
            else
            {
                DTW_SEC = d + MIN(DTW_FIR,MIN(DTW_SEC,DTW_DOWN));
            }

            bool ifcb = (tid == 16) && (j_temp == m/2);
            ifcb = __shfl_sync(0x1F, ifcb, 16);
            if(ifcb)
            {
                cb_index =(w + j_temp)/CB_LEN+1;
                cb_temp =cb[cb_index];
                vote = __ballot_sync(0xFFFFFFFF, DTW_SEC > threshold_2 - cb_temp);
                if (vote == 0xFFFFFFFF) {
                    if(tid == 16)
                    {
                        Dist = INFINITY;

                    }

                    return;
                    flag_pruning = true;
                }
            }

            i_temp++;

        }

    }

    if(!flag_pruning)
    {
        for(int i = 2*m  - 1- w;i < 2*m - 1;i++){
            mask = switch_for_stair%2;
            switch_for_stair++;
            
            if(!mask)
            {
                DTW_UP = __shfl_up_sync(0xFFFFFFFF, DTW_SEC, 1, 32);
                
                d = (i_temp < m && j_temp < m) ?  DIST(q[i_temp],t[j_temp]) : INFINITY;
                DTW_FIR = d + MIN(DTW_FIR,MIN(DTW_SEC,DTW_UP));

                j_temp++;
            }
                
            else
            {
                DTW_DOWN = __shfl_down_sync(0xFFFFFFFF, DTW_FIR, 1, 32);
                
                d = (i_temp < m && j_temp < m) ?  DIST(q[i_temp],t[j_temp]) : INFINITY;
                DTW_SEC = d + MIN(DTW_FIR,MIN(DTW_SEC,DTW_DOWN));

                i_temp++;
            }

        }

        if(tid == 16)
        {
            Dist = sqrt(DTW_FIR);

        }
    }

}

