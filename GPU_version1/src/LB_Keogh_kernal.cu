#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include "GPU_parameters.h"
#include "matrix.cuh"
__global__ void process_lbkeogh_kernel
        (FLOAT **my_subs, int *d_indices, int subseqLen, int diag_offset, FLOAT *bsf, int subcount,
         int diag, int *d_counter, int ran_idx, FLOAT **my_L, FLOAT **my_U, bool *lb_vector,
         FLOAT *cb1, FLOAT *cb2)
{
    
    int *indices_local = &d_indices[subcount*(diag_offset - diag)];
    bool *lb_local = &lb_vector[(diag_offset - diag)*subcount];
    
    FLOAT threshold = bsf[ran_idx];
    FLOAT threshold_2 = threshold * threshold;
    int bid = blockIdx.x;
    int tid = threadIdx.x%32;
    int step = (subseqLen/32);
    
    int d_counter_temp = d_counter[diag_offset - diag];
    for (int i = bid; i < d_counter_temp; i += gridDim.x ) {
        int original_index = indices_local[i];
        FLOAT*  sub1 = my_subs[original_index];
        FLOAT*  sub2 = my_subs[original_index + diag_offset - 1];
        FLOAT* U1 = my_U[original_index];
        FLOAT* L1 = my_L[original_index];
        FLOAT* U2 = my_U[original_index + diag_offset - 1];
        FLOAT* L2 = my_L[original_index + diag_offset - 1];

        FLOAT partial_dist1;
        partial_dist1 = lb_keogh_warp(sub1+ tid *step, U2 + tid*step, L2+ tid*step, step);
        FLOAT partial_dist2;
        partial_dist2 = lb_keogh_warp(sub2+ tid*step, U1+ tid*step, L1+ tid*step, step);

        #pragma unroll
        for (int mask = 1; mask <= 16; mask <<= 1) {
            
            FLOAT tmp1 = __shfl_down_sync(0xFFFFFFFF, partial_dist1, mask);
            FLOAT tmp2 = __shfl_down_sync(0xFFFFFFFF, partial_dist2, mask);
            if (tid + mask < 32) {
                partial_dist1 += tmp1;
                partial_dist2 += tmp2;
            }
        }
        cb1[tid] = partial_dist1;
        cb2[tid] = partial_dist2;

        if (tid == 0 && (partial_dist1 > threshold_2 || partial_dist2 > threshold_2)) {
              
            lb_local[i] = true;

        }
    }
}
