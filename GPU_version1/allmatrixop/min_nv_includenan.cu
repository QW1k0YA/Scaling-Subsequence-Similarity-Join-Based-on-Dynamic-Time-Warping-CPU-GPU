#include "hip/hip_runtime.h"

#include <iostream>
#include <vector>
#include <algorithm>
#include <cmath>
#include <limits>
#include "../alldef/typedefdouble.cuh"
#include "../alldef/elseoperation.cuh"
#include "../alldef/matrix.cuh"
using namespace std;
std::vector<FLOAT > min_nv_Includenan(FLOAT  x, const std::vector<FLOAT >& data) {

    std::vector<FLOAT > result;

    for (auto value : data)
    {
        if (isnan(value))
        {
            result.push_back(NAN);
        }
        else
        {
            result.push_back(MIN(x, value));
        }
    }

    return result;
}

std::vector<FLOAT > max_nv_Includenan(FLOAT  x, const std::vector<FLOAT >& data) {

    std::vector<FLOAT > result;

    for (auto value : data)
    {
        if (isnan(value))
        {
            result.push_back(NAN);
        }
        else
        {
            result.push_back(MAX(x, value));
        }
    }

    return result;
}
