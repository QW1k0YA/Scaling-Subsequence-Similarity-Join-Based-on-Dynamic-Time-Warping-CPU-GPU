#include "hip/hip_runtime.h"

#include <iostream>
#include <vector>
#include "../alldef/matrix.cuh"

#include "cmath"
using namespace std;

void mvmean(const std::vector<FLOAT > &a, int l, vector<FLOAT > &miu, vector<FLOAT > &si)
{
    size_t len_a = a.size();
    miu.reserve(len_a);
    si.reserve(len_a);
    FLOAT sum1 = 0;
    FLOAT sum2 = 0;
    for(int i = 0;i < l;i++)
    {
        sum1+= a[i];
        sum2+= a[i]*a[i];
    }

    long long ll = len_a - l +1;
    for(int i = 0;i < ll - 1;i++)
    {
        miu[i] = sum1/l;
        si[i] = sqrt(sum2/l-miu[i]*miu[i]);
        sum1 = sum1 - a[i] + a[i+l];
        sum2 = sum2 - a[i]*a[i] + a[i+l]*a[i+l];
    }
    miu[ll-1] = sum1/l;
    si[ll-1] = sqrt(sum2/l-miu[ll-1]*miu[ll-1]);

}
void mvmean_miu(const std::vector<FLOAT >& a, int len_a, int l, vector<FLOAT >& miu)
{

    FLOAT sum1 = 0;
    FLOAT sum2 = 0;
    for(int i = 0;i < l;i++)
    {
        sum1+= a[i];
    }

    long long ll = len_a - l +1;
    for(int i = 0;i < ll;i++)
    {
        miu[i] = sum1/l;
        sum1 = sum1 - a[i] + a[i+l];
    }

}

std::vector<FLOAT > movmean(const std::vector<FLOAT >& ts, int a, int b, bool c) {

    size_t ts_size = ts.size();
    if(c)
    {
        std::vector<FLOAT > result;

        if (ts.empty() || a < 0 || b < 0 || a + b == 0) {
            std::cerr << "Invalid input parameters. in movmean" << std::endl;
            return result;
        }

        for (size_t i = a; i < ts_size - b; ++i) {
            FLOAT  sum = 0.0;
            for (int j = -a; j <= b; ++j) {
                sum += ts[i + j];
            }
            result.push_back(sum / (a + b + 1));
        }

        return result;

    }
    else
    {
        std::vector<FLOAT > result;

        if (ts.empty() || a < 0 || b < 0 || a + b == 0) {
            std::cerr << "Invalid input parameters.in movmean" << std::endl;
            return result;
        }

        for (size_t i = 0; i < ts_size; ++i) {
            FLOAT  sum = 0.0;
            int count = 0;

            for (int j = -a; j <= b; ++j) {
                if (i + j >= 0 && i + j < ts_size) {
                    sum += ts[i + j];
                    count++;
                }
            }

            if (count > 0) {
                result.push_back(sum / count);
            }
            else {
                
                result.push_back(ts[i]);
            }
        }

        return result;
    }
}
