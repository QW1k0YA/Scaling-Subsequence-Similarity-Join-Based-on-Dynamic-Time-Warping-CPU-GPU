#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include "GPU_parameters.h"
#include "matrix.cuh"

__global__ void syn_bsfs(FLOAT *bsf)
{
    __shared__ FLOAT bsf_array[BSF_POOL];
    int tid = threadIdx.x;
    bsf_array[tid] = bsf[tid];

    for (int stride = blockDim.x/2; stride > 0; stride >>= 1) {
        if (tid < stride && tid + stride < BSF_POOL) {
            bsf_array[tid] = fminf(bsf_array[tid],bsf_array[tid + stride]);
        }
        __syncthreads();
    }

    bsf[tid] = bsf_array[0];

}