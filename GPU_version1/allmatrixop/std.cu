#include "hip/hip_runtime.h"

#include <iostream>
#include <vector>
#include <cmath>
#include "../alldef/matrix.cuh"
FLOAT  stddev(const std::vector<FLOAT >& data) {

    FLOAT  mean = 0.0;
    for (FLOAT  value : data) {
        mean += value;
    }
    mean /= data.size();

    FLOAT  sumSquaredDifferences = 0.0;
    for (FLOAT  value : data) {
        sumSquaredDifferences += (value - mean) * (value - mean);
    }

    FLOAT  variance = sumSquaredDifferences / (data.size());
    
    FLOAT  standardDeviation = std::sqrt(variance);

    return standardDeviation;
}