#include "hip/hip_runtime.h"

#include <iostream>
#include <vector>
#include <cmath>
#include "../alldef/typedefdouble.cuh"
std::vector<bool> isNaN(const std::vector<FLOAT >& vec) {
    std::vector<bool> result;

    for (FLOAT  value : vec) {
        result.push_back(std::isnan(value));
    }

    return result;
}

