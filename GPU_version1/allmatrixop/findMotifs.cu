#include "hip/hip_runtime.h"

#include "../alldef/matrix.cuh"
#include "iostream"
#include "vector"
#include "algorithm"
#include "cmath"
#include "chrono"
#include "../alldef/allstruct.cuh"
#include "../alldef/typedefdouble.cuh"
#include "../allunder/undermpx_v2.cuh"
#include "numeric"
#include <thread>
#include "../allunder/undermpx_v2.cuh"
#include <functional>

using namespace std;

void findMotifs(const vector<FLOAT >& timeSeries, const vector<FLOAT >& mu, const vector<FLOAT >& invnorm,
                const vector<FLOAT >& matrixProfile, const vector<int> &profileIndex, int subseqLen, int exclusionLen, vector <vector<int>> &result)
{
    FLOAT  motifCount = 3;
    FLOAT  radius = 2;
    FLOAT  neighborCount = 10;
    vector<FLOAT > matrixProfile_ = matrixProfile;

    int padLen;

    std::function< vector<FLOAT >(int)> crosscov_;
    if(subseqLen < thread::hardware_concurrency()*128 )
    {
        auto crosscov = [=](int idx) -> vector<FLOAT >{
            vector<FLOAT > temp1,temp2,temp3,temp4;
            for(int i = idx + subseqLen -1;i >= idx ;i--)
            {
                temp1.push_back(timeSeries[i-1] - mu[idx-1]);
            }

            temp2 = elementwiseMultiply_nv(invnorm[idx-1],temp1);
            return convolve_valid(timeSeries,temp2);
        };
        crosscov_ = crosscov;
    }
    else
    {
        padLen = pow(2, nextpow2(timeSeries.size()));
        auto crosscov = [=](int idx) -> vector<FLOAT >{
            vector<FLOAT > temp1,temp2;
            vector<complex<FLOAT >> temp3,temp4,temp5,temp6;
            for(int i = idx;i <= idx + subseqLen -1 ;i++)
            {
                temp1.push_back(timeSeries[i-1] - mu[idx-1]);
            }
            temp2 = elementwiseMultiply_nv(invnorm[idx-1],temp1);

            temp3 = fft(temp2,padLen);
            temp4 = fft(timeSeries,padLen);

            temp5 = conjugate(temp4);

            temp6 = elementWiseMultiply_complex(temp3,temp4);
            return ifft(temp6, true);

        };
        crosscov_ = crosscov;
    }

    FLOAT  corr = 0;
    int motIdx = 1;
    for(int i = 1;i <= motifCount;i++)
    {

        int j = 1;
        corr = matrixProfile[j-1];

        for(auto value:matrixProfile)
        {
            if(value > corr || (isnan(corr)&&(!isnan(value))))
            {
                corr = value;
                motIdx = j;
            }
            j++;
        }

        vector<FLOAT > corr_(1, corr);
        FLOAT  exclRangeBegin,exclRangeEnd;

        if((isinfinite(corr_)[0]) || (abs(corr + 1) < EPS))
        {

            break;
        }

        result[0][i - 1] = MIN(motIdx, profileIndex[motIdx - 1]);
        result[1][i - 1] = max(motIdx, profileIndex[motIdx - 1]);

        auto corrProfile = crosscov_(motIdx);
        corrProfile = min_nv_Includenan(1, elementWiseMultiply(
                extr_vfromv(corrProfile, 1, timeSeries.size() - subseqLen + 1), invnorm ));

        size_t matrixProfile_size = matrixProfile.size();
        for(int k = 0;k < matrixProfile_size;k++)
        {
            if(isnan(matrixProfile[k]))
            {
                corrProfile[k] = NAN;
            }
        }

        if(exclusionLen > 0)
        {
            for(int j = 1;j <= 2;j++)
            {
                exclRangeBegin = max(1, result[j - 1][i - 1] - exclusionLen + 1);

                exclRangeEnd = MIN(FLOAT (matrixProfile.size()), result[j - 1][i - 1] + exclusionLen - 1);

                for(int l = exclRangeBegin;l <= exclRangeEnd;l++)
                {
                    corrProfile[l-1] = NAN;
                }

            }
        }

        FLOAT  neighborCorr = corrProfile[0];
        int neighbor = 0;

        size_t corrProfile_size = corrProfile.size();

        for(int j = 3;j <= neighborCount + 2;j++)
        {

            neighborCorr = corrProfile[0];
            neighbor = 1;

            for(int o = 1;o <= corrProfile_size;o++)
            {

                if(corrProfile[o-1] > neighborCorr || (isnan(neighborCorr) && !isnan(corrProfile[o-1])) )
                {
                    neighborCorr = corrProfile[o-1];
                    neighbor = o;
                }

            }
            
            vector<FLOAT > neighborCorr_v(1, neighborCorr);

            if(isinfinite(neighborCorr_v)[0] || ((1 - neighborCorr) >= (radius * (1 - corr_[0]))))
            {

                break;
            }

            result[j - 1][i - 1] = neighbor;

            if(exclusionLen > 0)
            {
                exclRangeBegin = max(1.0, FLOAT (neighbor - exclusionLen + 1));
                exclRangeEnd = MIN(FLOAT (matrixProfile.size()), FLOAT (neighbor + exclusionLen - 1));

                for(int l = exclRangeBegin;l <= exclRangeEnd;l++)
                {
                    corrProfile[l-1] = NAN;
                }
            }
        }

        for(int k = 0;k < corrProfile_size;k++)
        {
            if(isnan(corrProfile[k]))
            {

                matrixProfile_[k] = NAN;
            }
        }

    }

    result = result;
}

