#include "hip/hip_runtime.h"

#include "iostream"
#include "vector"
#include "../alldef/matrix.cuh"
#include "cmath"

using namespace std;

FLOAT  norm_vector(const vector<FLOAT >& A, int b)
{
    if (b == 2)
    {
        FLOAT  sum = 0;
        size_t A_Size = A.size();
        for (int i = 0; i < A_Size; i++)
        {
            sum += pow(A[i], 2);
        }
        sum = pow(sum, 0.5);
        return sum;
    }
}