#include "hip/hip_runtime.h"

#include <iostream>
#include <vector>
#include <cmath>
#include "../alldef/typedefdouble.cuh"
std::vector<bool> isinfinite(const std::vector<FLOAT >& vec) {
    std::vector<bool> result;

    for (FLOAT  value : vec) {
        result.push_back(!std::isfinite(value));
    }

    return result;
}